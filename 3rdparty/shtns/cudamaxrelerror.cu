#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "fftw3.h"
#include "shtns.h"
#include "shtns_cuda.h"

template <typename T1, typename T2> T1 maxrelerr(T1 *expected, T2 *actual, unsigned int n) {
    T1 mre = 0.0f;
    unsigned int imax = 0;
    for (unsigned int i=0; i<n; i++)
    {
        T1 re = abs(expected[i] - actual[i]) / abs(expected[i]);
        if (i < 10)
            printf("re[%d] %g <- (%g, %g)\n", i, re, expected[i], actual[i]);
        if (re > mre) { mre = re; imax=i; }
    }
    printf("max re[%d] %g <- (%g, %g)\n", imax, mre, expected[imax], actual[imax]);
    return mre;
}

double test_double_host(int nlon, int nlat, int lmax, double *init_x) {
    int nspat = nlon * nlat;
    double *x1, *x2;
    cplx *q;
    shtns_cfg sht = shtns_create(lmax, lmax, 1, sht_orthonormal);
    shtns_set_grid(sht, ((enum shtns_type)(sht_quick_init)), 0, nlat, nlon);
    shtns_print_cfg(sht);
    x1 = (double*) fftw_malloc(nspat * sizeof(double));
    x2 = (double*) fftw_malloc(nspat * sizeof(double));
    q = (cplx*) fftw_malloc(sht->nlm * sizeof(cplx));
    memcpy(x1, init_x, nspat * sizeof(double));
//    memset(x2, 0, nspat * sizeof(double));
    spat_to_SH(sht, x1, q);
    SH_to_spat(sht, q, x2);
	memcpy(init_x, x2, nspat * sizeof(double));
    spat_to_SH(sht, x2, q);
    SH_to_spat(sht, q, x1);
    //return maxrelerr(x2, x1, nspat);
    return maxrelerr(init_x, x1, nspat);
}

double test_double_gpu(int nlon, int nlat, int lmax, double *init_x) {
    int nspat = nlon * nlat;
    double *x1, *x2, *hx1, *hx2;
    cplx *q;
    shtns_cfg sht = shtns_create(lmax, lmax, 1, sht_orthonormal);
    shtns_set_grid(sht, ((enum shtns_type)(sht_quick_init|SHT_ALLOW_GPU)), 0, nlat, nlon);
    shtns_print_cfg(sht);
    hipMalloc(&x1, nspat * sizeof(double));
    hipMalloc(&x2, nspat * sizeof(double));
    hipMalloc(&q, sht->nlm * sizeof(cplx));
    hipMemcpy(x1, init_x, nspat * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(x2, 0, nspat * sizeof(double));

    hx1 = (double*) malloc(nspat * sizeof(double));
    hx2 = (double*) malloc(nspat * sizeof(double));

    cu_spat_to_SH(sht, x1, q, lmax);
    cu_SH_to_spat(sht, q, x2, lmax);
    	hipMemcpy(hx2, x2, nspat * sizeof(double), hipMemcpyDeviceToHost);
    cu_spat_to_SH(sht, x2, q, lmax);
    cu_SH_to_spat(sht, q, x1, lmax);

    hipMemcpy(hx1, x1, nspat * sizeof(double), hipMemcpyDeviceToHost);
    maxrelerr(hx2, hx1, nspat);               // error for back & forth within GPU
    return maxrelerr(init_x, hx1, nspat);     // error comparing GPU with CPU (assuming init_x is the CPU data)
}

//double test_float() {
//    cu_spat_to_SH_float(sht, c_x64, c_Qlm, lmax);
//    cu_SH_to_spat_float(sht, c_Qlm, c_x64, lmax);
//}

int main()
{
    int nlon=520, nlat=256, lmax=150;
    double *init_x = (double*) malloc(nlat*nlon*sizeof(double));

//    shtns_verbose(1);
//    shtns_print_version();
    shtns_use_gpu(0);

    for (int i=0; i<(nlat*nlon); i++)
        init_x[i] = (1.0 * rand()) / RAND_MAX - 0.5;

    printf("TEST HOST\n");
    test_double_host(nlon, nlat, lmax, init_x);
    printf("TEST GPU\n");
    test_double_gpu(nlon, nlat, lmax, init_x);
//    double float_max_relerr = test_float(nlon, nlat, lmax, init_x);

    return 0;
}
