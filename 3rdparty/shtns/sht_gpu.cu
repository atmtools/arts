#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2021 Centre National de la Recherche Scientifique.
 * written by Nathanael Schaeffer (CNRS, ISTerre, Grenoble, France).
 * 
 * nathanael.schaeffer@univ-grenoble-alpes.fr
 * 
 * This software is governed by the CeCILL license under French law and
 * abiding by the rules of distribution of free software. You can use,
 * modify and/or redistribute the software under the terms of the CeCILL
 * license as circulated by CEA, CNRS and INRIA at the following URL
 * "http://www.cecill.info".
 * 
 * The fact that you are presently reading this means that you have had
 * knowledge of the CeCILL license and that you accept its terms.
 * 
 */

/* NOTES:
 * - the cuda transforms are NOT thread-safe. Use cushtns_clone() to clone transforms for each thread.
*/

/* TODO
 * 0) DYNAMIC THREADS/BLOCK, computed on init.
 * 1) use static polar optimization (from constant memory ?)
 * 2) use a for loop in m-direction to re-use threads at larger m's ?
 * 3) implement FFT on host versions. otpimize data transfer (don't transfer zeros): store as complex2real fft ?
 * 4) allow several variants, which may change occupancy for large sizes ?
 * 5) generalize NFIELDS to all kernels
 * 6) find optimal threads/block for minor kernels too (e.g. sphtor2scal)
 * 7) try to use transposed fft, to see if it is faster than my own FFT + transpose...   => No, it's not.
 */

/* Session with S. Chauveau from nvidia:
 * useful metrics = achieved_occupancy, cache_hit
 * for leg_m_lowllim, the "while(l<llim)" loop:
 * 		0) full al, ql load before the while loop.	=> DONE
 * 		1) reduce pointer update by moving ql and al updates into the "if" statement.	=> DONE
 * 	    2a) try to use a double-buffer (indexed by b, switched by b=1-b)		=> only 1 __syncthread() instead of 2.
 * 	OR:	2b) preload al, ql into registers => may reduce the waiting at __syncthreads()
 * 		3) unroll by hand the while loop (with an inner-loop of fixed size)		=> DONE
 * 		4) introduce NWAY (1 to 4) to avoid the need of several blocks in theta => one block for all means al, ql are read only once!	=> DONE
 * 				=> increases register pressure, but may be OK !
 */

// NOTE variables gridDim.x, blockIdx.x, blockDim.x, threadIdx.x, and warpSize are defined in device functions
/* NOTE:
 * 				KEPLER							PASCAL
 * cache-line:  128 bytes (16 doubles)
 * 
 * fetching 1 double/thread: 2 requests/warp
 */

#include "sht_private.h"

/// Maximum number of threads per block that should be used.
#define MAX_THREADS_PER_BLOCK 512
/// The warp size is always 32 on cuda devices (up to Pascal at least)
#define WARPSZE 32

#include "sht_gpu_kernels.cu"

enum cushtns_flags { CUSHT_OFF=0, CUSHT_ON=1, CUSHT_OWN_COMP_STREAM=2, CUSHT_OWN_XFER_STREAM=4};
enum cushtns_fft_mode { CUSHT_NOFFT, CUSHT_FFT_THETA_CONTIGUOUS, CUSHT_FFT_TRANSPOSE };

/* TOOL FUNCTIONS */

extern "C"
void* shtns_malloc(size_t size) {
	void* ptr = NULL;
	hipError_t err = hipSuccess;
	err = hipHostMalloc(&ptr, size);		// try to allocate pinned memory (for faster transfers !)
	if (err != hipSuccess) {
		ptr = VMALLOC(size);		// return regular memory instead...
	}
	return ptr;
}

extern "C"
void shtns_free(void* p) {
	hipHostFree(p);
}

void memzero_omp(double* mem, const size_t sze)
{
	#ifdef _OPENMP
	#pragma omp parallel
	{
		int i = omp_get_thread_num();
		int n = omp_get_num_threads();
		int ofs = (i*sze)/n;
		memset(mem + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
	}
	#else
		memset(mem, 0, sze*sizeof(double));
	#endif
}

void memzero_omp(double* mem, double* mem2, const size_t sze)
{
	#ifdef _OPENMP
	#pragma omp parallel
	{
		int i = omp_get_thread_num();
		int n = omp_get_num_threads();
		int ofs = (i*sze)/n;
		memset(mem + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
		memset(mem2 + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
	}
	#else
		memset(mem,  0, sze*sizeof(double));
		memset(mem2, 0, sze*sizeof(double));
	#endif
}

void memzero_omp(double* mem, double* mem2, double* mem3, const size_t sze)
{
	#ifdef _OPENMP
	#pragma omp parallel
	{
		int i = omp_get_thread_num();
		int n = omp_get_num_threads();
		int ofs = (i*sze)/n;
		memset(mem + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
		memset(mem2 + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
		memset(mem3 + ofs, 0, ((i+1)*sze/n - ofs)*sizeof(double));
	}
	#else
		memset(mem,  0, sze*sizeof(double));
		memset(mem2, 0, sze*sizeof(double));
		memset(mem3, 0, sze*sizeof(double));
	#endif
}



extern "C"
void cushtns_release_gpu(shtns_cfg shtns)
{
	// TODO: arrays possibly shared between different shtns_cfg should be deallocated ONLY if not used by other shtns_cfg.
	if (shtns->nphi > 1) hipfftDestroy(shtns->cufft_plan);
	if (shtns->nphi > 1) hipfftDestroy(shtns->cufft_plan_float);
	if (shtns->cu_flags & CUSHT_OWN_COMP_STREAM) hipStreamDestroy(shtns->comp_stream);
	if (shtns->cu_flags & CUSHT_OWN_XFER_STREAM) hipStreamDestroy(shtns->xfer_stream);
	if (shtns->d_ct) hipFree(shtns->d_ct);
	if (shtns->d_ct_f) hipFree(shtns->d_ct_f);
	if (shtns->d_alm) hipFree(shtns->d_alm);
	if (shtns->d_alm_f) hipFree(shtns->d_alm_f);
	if (shtns->d_mx_stdt) hipFree(shtns->d_mx_stdt);
	if (shtns->d_mx_van) hipFree(shtns->d_mx_van);
	if (shtns->gpu_mem) hipFree(shtns->gpu_mem);
	if (shtns->xfft) hipFree(shtns->xfft);
	if (shtns->xfft_cpu) shtns_free(shtns->xfft_cpu);
	shtns->d_alm = 0;		// disable gpu.
	shtns->cu_flags = 0;
}

static int init_cuda_buffer_fft(shtns_cfg shtns)
{
	hipError_t err = hipSuccess;
	int err_count = 0;
	const unsigned layout = shtns->layout & (256*7);	// isolate layout

	shtns->comp_stream = 0;		// use default stream for computations.
	shtns->cu_flags &= ~((int)CUSHT_OWN_COMP_STREAM);		// mark the compute stream (=default stream) as NOT managed by shtns.
	err = hipStreamCreateWithFlags(&shtns->xfer_stream, hipStreamNonBlocking);		// stream for async data transfer.
	shtns->cu_flags |= CUSHT_OWN_XFER_STREAM;		// mark the transfer stream as managed by shtns.
	if (err != hipSuccess)  err_count ++;

	/* cuFFT init */
	int nfft = shtns->nphi;
	//int nreal = 2*(nfft/2+1);
	shtns->cu_fft_mode = CUSHT_NOFFT;
	if (nfft > 1) {
		// hipfftPlanMany(hipfftHandle *plan, int rank, int *n,   int *inembed, int istride, int idist,   int *onembed, int ostride, int odist,   hipfftType type, int batch);
		hipfftResult res, res_float;
		if (layout == SHT_PHI_CONTIGUOUS) {
			printf("WARNING: phi-contiguous transform not available on GPU.\n");
			err_count ++;
			return 1;
		} else if ((layout == SHT_NATIVE_LAYOUT) && (nfft % 16 == 0) && (shtns->nlat_2 % 16 == 0)) {	// use the fastest data-layout.
			printf("!!! Use phi-contiguous FFT +transpose: WARNING, the spatial data is neither phi-contiguous nor theta-contiguous !!!\n");
			res = hipfftPlanMany(&shtns->cufft_plan, 1, &nfft, &nfft, 1, shtns->nphi, &nfft, 1, shtns->nphi, HIPFFT_Z2Z, shtns->nlat_2);
			res_float = hipfftPlanMany(&shtns->cufft_plan_float, 1, &nfft, &nfft, 1, shtns->nphi, &nfft, 1, shtns->nphi, HIPFFT_C2C, shtns->nlat_2);
			shtns->cu_fft_mode = CUSHT_FFT_TRANSPOSE;
			//hipfftPlanMany(&shtns->cufft_plan, 1, &nfft, &nfft, 1, shtns->nphi, &nreal, 1, shtns->nphi, HIPFFT_D2Z, shtns->nlat);
		} else {		// if (layout & SHT_THETA_CONTIGUOUS)
			printf("have a theta contiguous layout\n");
			res = hipfftPlanMany(&shtns->cufft_plan, 1, &nfft, &nfft, shtns->nlat_2, 1, &nfft, shtns->nlat_2, 1, HIPFFT_Z2Z, shtns->nlat_2);
			res_float = hipfftPlanMany(&shtns->cufft_plan_float, 1, &nfft, &nfft, shtns->nlat_2, 1, &nfft, shtns->nlat_2, 1, HIPFFT_C2C, shtns->nlat_2);
			shtns->cu_fft_mode = CUSHT_FFT_THETA_CONTIGUOUS;
		}
		if (res != HIPFFT_SUCCESS || res_float != HIPFFT_SUCCESS) {
			printf("cufft init FAILED!\n");
			err_count ++;
		}
		size_t worksize;
		hipfftGetSize(shtns->cufft_plan, &worksize);
		printf("work-area size: %ld \t nlat*nphi = %ld\n", worksize/8, shtns->spat_stride);
		hipfftGetSize(shtns->cufft_plan_float, &worksize);
		printf("float work-area size: %ld \t nlat*nphi = %ld\n", worksize/4, shtns->spat_stride);
	}

	// Allocate working arrays for SHT on GPU:
	double* gpu_mem = NULL;
	const int nlm2 = shtns->nlm + (shtns->mmax+1);		// one more data per m
	const size_t nlm_stride = ((2*nlm2+WARPSZE-1)/WARPSZE) * WARPSZE;
	const size_t spat_stride = ((shtns->nlat*shtns->nphi+WARPSZE-1)/WARPSZE) * WARPSZE;
	const size_t dual_stride = (spat_stride < nlm_stride) ? nlm_stride : spat_stride;		// we need two spatial buffers to also hold spectral data.
	if (shtns->cu_fft_mode == CUSHT_FFT_TRANSPOSE) {
		hipMalloc( (void **)&shtns->xfft, spat_stride * sizeof(double));
	}
	err = hipMalloc( (void **)&gpu_mem, (2*nlm_stride + 2*dual_stride + spat_stride)*sizeof(double) );		// maximum GPU memory required for SHT
	if (err != hipSuccess)	err_count++;
	
	if (shtns->fftc_mode > 0) {
		// we also need a buffer on the CPU when the FFT is out-of-place:
		shtns->xfft_cpu = (double*) shtns_malloc(spat_stride * sizeof(double));
	}

	shtns->nlm_stride = nlm_stride;
	shtns->spat_stride = dual_stride;
	shtns->gpu_mem = gpu_mem;

	return err_count;
}

extern "C"
int cushtns_init_gpu(shtns_cfg shtns)
{
	hipError_t err = hipSuccess;
	const long nlm = shtns->nlm;
	const long nlat_2 = shtns->nlat_2;

	double *d_alm = NULL;
	float *d_alm_f = NULL;
	double *d_ct  = NULL;
	float *d_ct_f  = NULL;
	double *d_mx_stdt = NULL;
	double *d_mx_van = NULL;
	int err_count = 0;
	int device_id = -1;

	hipDeviceProp_t prop;
	hipGetDevice(&device_id);
	err = hipGetDeviceProperties(&prop, device_id);
	if (err != hipSuccess) return -1;
	#if SHT_VERBOSE > 0
	printf("  cuda GPU #%d \"%s\" found (warp size = %d, compute capabilities = %d.%d).\n", device_id, prop.name, prop.warpSize, prop.major, prop.minor);
	#endif
	if (prop.warpSize != WARPSZE) return -1;		// failure, SHTns requires a warpSize of 32.
	if (prop.major < 3) return -1;			// failure, SHTns requires compute cap. >= 3 (warp shuffle instructions)

	// Allocate the device input vector alm
	err = hipMalloc((void **)&d_alm, (2*nlm+MAX_THREADS_PER_BLOCK-1)*sizeof(double));	// allow some overflow.
	if (err != hipSuccess) err_count ++;
	err = hipMalloc((void **)&d_alm_f, (2*nlm+MAX_THREADS_PER_BLOCK-1)*sizeof(float));	// allow some overflow.
	if (err != hipSuccess) err_count ++;
	if (shtns->mx_stdt) {
		// Allocate the device matrix for d(sin(t))/dt
		err = hipMalloc((void **)&d_mx_stdt, (2*nlm+MAX_THREADS_PER_BLOCK-1)*sizeof(double));
		if (err != hipSuccess) err_count ++;
		// Same thing for analysis
		err = hipMalloc((void **)&d_mx_van, (2*nlm+MAX_THREADS_PER_BLOCK-1)*sizeof(double));
		if (err != hipSuccess) err_count ++;
	}
	// Allocate the device input vector cos(theta) and gauss weights
	err = hipMalloc((void **)&d_ct, 2*nlat_2*sizeof(double));
	if (err != hipSuccess) err_count ++;
	err = hipMalloc((void **)&d_ct_f, 2*nlat_2*sizeof(float));
	if (err != hipSuccess) err_count ++;

	if (err_count == 0) {
		err = hipMemcpy(d_alm, shtns->alm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
		if (err != hipSuccess)  err_count ++;
		if (shtns->mx_stdt) {
			err = hipMemcpy(d_mx_stdt, shtns->mx_stdt, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
			if (err != hipSuccess)  err_count ++;
			err = hipMemcpy(d_mx_van, shtns->mx_van, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
			if (err != hipSuccess)  err_count ++;
		}
		err = hipMemcpy(d_ct, shtns->ct, nlat_2*sizeof(double), hipMemcpyHostToDevice);
		if (err != hipSuccess)  err_count ++;
		err = hipMemcpy(d_ct + nlat_2, shtns->wg, nlat_2*sizeof(double), hipMemcpyHostToDevice);
		if (err != hipSuccess)  err_count ++;
	}

	shtns->d_alm = d_alm;
	shtns->d_alm_f = d_alm_f;
	shtns->d_ct  = d_ct;
	shtns->d_ct_f  = d_ct_f;
	shtns->d_mx_stdt = d_mx_stdt;
	shtns->d_mx_van = d_mx_van;
	copy_convert(shtns);

	err_count += init_cuda_buffer_fft(shtns);

	if (err_count != 0) {
		cushtns_release_gpu(shtns);
		return -1;	// fail
	}

	return device_id;		// success, return device_id
}

/// \internal Enables parallel transforms on selected GPU device, if available. \see shtns_use_gpu
extern "C"
int cushtns_use_gpu(int device_id)
{
	int count = 0;
	if (device_id >= 0) {
		hipGetDeviceCount(&count);
		if (count > 0) {
			device_id = device_id % count;
			hipSetDevice(device_id);
			return device_id;
		}
	}
	return -1;		// disable gpu.
}

extern "C"
void cushtns_set_streams(shtns_cfg shtns, hipStream_t compute_stream, hipStream_t transfer_stream)
{
	if (compute_stream != 0) {
		if (shtns->cu_flags & CUSHT_OWN_COMP_STREAM) hipStreamDestroy(shtns->comp_stream);
		shtns->comp_stream = compute_stream;
		if (shtns->nphi > 1) hipfftSetStream(shtns->cufft_plan, compute_stream);
		if (shtns->nphi > 1) hipfftSetStream(shtns->cufft_plan_float, compute_stream);
		shtns->cu_flags &= ~((int)CUSHT_OWN_COMP_STREAM);		// we don't manage this stream
	}
	if (transfer_stream != 0) {
		if (shtns->cu_flags & CUSHT_OWN_XFER_STREAM) hipStreamDestroy(shtns->xfer_stream);
		shtns->xfer_stream = transfer_stream;
		shtns->cu_flags &= ~((int)CUSHT_OWN_XFER_STREAM);		// we don't manage this stream
	}
}

/*
extern "C"
shtns_cfg cushtns_clone(shtns_cfg shtns, hipStream_t compute_stream, hipStream_t transfer_stream)
{
	if (shtns->d_alm == 0) return 0;		// do not clone if there is no GPU associated...

	shtns_cfg sht_clone;
	sht_clone = shtns_create_with_grid(shtns, shtns->mmax, 0);		// copy the shtns_cfg, sharing all data.

	// set new buffer and cufft plan (should be unique for each shtns_cfg).
	int err_count = init_cuda_buffer_fft(sht_clone);
	if (err_count > 0) return 0;		// TODO: memory should be properly deallocated here...
	// set new streams (should also be unique).
	cushtns_set_streams(sht_clone, compute_stream, transfer_stream);
	return sht_clone;
}
*/

extern "C"
shtns_cfg cushtns_clone(shtns_cfg shtns, hipStream_t compute_stream, hipStream_t transfer_stream)
{
	shtns_cfg sht_clone;
	sht_clone = shtns_create_with_grid(shtns, shtns->mmax, 0);		// copy the shtns_cfg, sharing all data.

	int dev_id = cushtns_init_gpu(sht_clone);
	if (dev_id >= 0) {
		cushtns_set_streams(sht_clone, compute_stream, transfer_stream);
		return sht_clone;
	} else {
		shtns_destroy(sht_clone);
		return 0;		// fail
	}
}

void fourier_to_spat_gpu(shtns_cfg shtns, double* q, const int mmax)
{
	const int nphi = shtns->nphi;
	hipfftResult res;
	if (nphi > 1) {
		hipfftDoubleComplex* x = (hipfftDoubleComplex*) q;
		if (shtns->cu_fft_mode == CUSHT_FFT_TRANSPOSE) {
			double* xfft = shtns->xfft;
			transpose_cplx_zero(shtns->comp_stream, (double*) x, xfft, shtns->nlat_2, nphi, mmax);		// zero out m>mmax during transpose
			res = hipfftExecZ2Z(shtns->cufft_plan, (hipfftDoubleComplex*) xfft, x, HIPFFT_BACKWARD);
		} else {	// THETA_CONTIGUOUS:
			if (2*(mmax+1) <= nphi) {
				const int nlat = shtns->nlat;
				hipMemsetAsync( q + (mmax+1)*nlat, 0, sizeof(double)*(nphi-2*mmax-1)*nlat, shtns->comp_stream );		// zero out m>mmax before fft
			}
			res = hipfftExecZ2Z(shtns->cufft_plan, x, x, HIPFFT_BACKWARD);
		}
		if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);
	}
}

void fourier_to_spat_gpu(shtns_cfg shtns, float* q, const int mmax)
{
	const int nphi = shtns->nphi;
	hipfftResult res;
	if (nphi > 1) {
		hipfftComplex* x = (hipfftComplex*) q;
		if (shtns->cu_fft_mode == CUSHT_FFT_TRANSPOSE) {
			float* xfft = (float*)shtns->xfft;
			transpose_cplx_zero(shtns->comp_stream, (float*) x, xfft, shtns->nlat_2, nphi, mmax);		// zero out m>mmax during transpose
			res = hipfftExecC2C(shtns->cufft_plan_float, (hipfftComplex*) xfft, x, HIPFFT_BACKWARD);
		} else {	// THETA_CONTIGUOUS:
			if (2*(mmax+1) <= nphi) {
				const int nlat = shtns->nlat;
				hipMemsetAsync( q + (mmax+1)*nlat, 0, sizeof(float)*(nphi-2*mmax-1)*nlat, shtns->comp_stream );		// zero out m>mmax before fft
			}
			res = hipfftExecC2C(shtns->cufft_plan_float, x, x, HIPFFT_BACKWARD);
		}
		if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);
	}
}

void spat_to_fourier_gpu(shtns_cfg shtns, double* q, const int mmax)
{
    const int nphi = shtns->nphi;
    hipfftResult res;
    if (nphi > 1) {
        hipfftDoubleComplex *x = (hipfftDoubleComplex*) q;
        if (shtns->cu_fft_mode == CUSHT_FFT_TRANSPOSE) {
            double* xfft = shtns->xfft;
            res = hipfftExecZ2Z(shtns->cufft_plan, x, (hipfftDoubleComplex*) xfft, HIPFFT_BACKWARD);
            transpose_cplx_skip(shtns->comp_stream, xfft, (double*) x, nphi, shtns->nlat_2, mmax);		// ignore m > mmax during transpose
        } else {	// THETA_CONTIGUOUS:
            res = hipfftExecZ2Z(shtns->cufft_plan, x, x, HIPFFT_BACKWARD);
        }
        if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);
    }
}

void spat_to_fourier_gpu(shtns_cfg shtns, float* q, const int mmax)
{
	const int nphi = shtns->nphi;
	hipfftResult res;
	if (nphi > 1) {
		hipfftComplex *x = (hipfftComplex*) q;
		if (shtns->cu_fft_mode == CUSHT_FFT_TRANSPOSE) {
			float* xfft = (float*)shtns->xfft;
			res = hipfftExecC2C(shtns->cufft_plan_float, x, (hipfftComplex*) xfft, HIPFFT_BACKWARD);
			transpose_cplx_skip(shtns->comp_stream, xfft, (float*) x, nphi, shtns->nlat_2, mmax);		// ignore m > mmax during transpose
		} else {	// THETA_CONTIGUOUS:
			res = hipfftExecC2C(shtns->cufft_plan_float, x, x, HIPFFT_BACKWARD);
		}
		if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);
	}
}

void spat_to_fourier_host(shtns_cfg shtns, double* q, double* qf)
{
	// FFT on host
	if (shtns->fftc_mode >= 0) {
		if (shtns->fftc_mode != 1) {
			fftw_execute_dft(shtns->fftc, (fftw_complex *) q, (fftw_complex *) qf);
		} else {		// split dft
			printf("ERROR fft not supported\n");
		}
	}
}

void fourier_to_spat_host(shtns_cfg shtns, double* qf, double* q)
{
	if (shtns->fftc_mode >= 0) {
		if (shtns->fftc_mode != 1) {
			fftw_execute_dft(shtns->ifftc, (fftw_complex *) qf, (fftw_complex *) q);
		} else {		// split dft
			printf("ERROR fft not supported\n");
		}
	}
}

/************************
 * TRANSFORMS ON DEVICE *
 ************************/ 


/// Perform SH transform on data that is already on the GPU. d_Qlm and d_Vr are pointers to GPU memory (obtained by hipMalloc() for instance)
template<int S, int NFIELDS, typename real=double>
void cuda_SH_to_spat(shtns_cfg shtns, std::complex<real>* d_Qlm, real* d_Vr, const long int llim, const int mmax, int spat_dist = 0)
{
    if (spat_dist == 0) spat_dist = shtns->spat_stride;
    legendre<S,NFIELDS,real>(shtns, (real*) d_Qlm, d_Vr, llim, mmax, spat_dist);
    for (int f=0; f<NFIELDS; f++)  fourier_to_spat_gpu(shtns, d_Vr + f*spat_dist, mmax);
}

/// Perform SH transform on data that is already on the GPU. d_Qlm and d_Vr are pointers to GPU memory (obtained by hipMalloc() for instance)
template<int S, int NFIELDS, typename real=double>
void cuda_spat_to_SH(shtns_cfg shtns, real* d_Vr, std::complex<real>* d_Qlm, const long int llim, int spat_dist = 0)
{
    int mmax = shtns->mmax;
    if (spat_dist == 0) spat_dist = shtns->spat_stride;
    for (int f=0; f<NFIELDS; f++) spat_to_fourier_gpu(shtns, d_Vr + f*spat_dist, mmax);
    ilegendre<S, NFIELDS,real>(shtns, d_Vr, (real*) d_Qlm, llim, spat_dist);
}

extern "C"
void cu_SH_to_spat(shtns_cfg shtns, cplx* d_Qlm, double *d_Vr, int llim)
{
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	if (llim < mmax*mres)	mmax = llim / mres;	// truncate mmax too !
	cuda_SH_to_spat<0,1>(shtns, d_Qlm, d_Vr, llim, mmax);
}

extern "C"
void cu_SH_to_spat_float(shtns_cfg shtns, cplx_f* d_Qlm, float *d_Vr, int llim)
{
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	if (llim < mmax*mres)	mmax = llim / mres;	// truncate mmax too !
	cuda_SH_to_spat<0,1,float>(shtns, d_Qlm, d_Vr, llim, mmax);
}

extern "C"
void cu_SHsphtor_to_spat(shtns_cfg shtns, cplx* d_Slm, cplx* d_Tlm, double* d_Vt, double* d_Vp, int llim)
{
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const long nlm_stride = shtns->nlm_stride;
	double* d_vwlm = shtns->gpu_mem;

	if (llim < mmax*mres)	mmax = llim / mres;	// truncate mmax too !

	sphtor2scal_gpu(shtns, d_Slm, d_Tlm, (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), llim, mmax);
	// SHT on the GPU
//	cuda_SH_to_spat<1,1>(shtns, (cplx*) d_vwlm, d_Vt, llim+1);
//	cuda_SH_to_spat<1,1>(shtns, (cplx*) (d_vwlm + nlm_stride), d_Vp, llim+1);
	cuda_SH_to_spat<1,2>(shtns, (cplx*) d_vwlm, d_Vt, llim+1, d_Vp-d_Vt, mmax);
}

extern "C"
void cu_SHqst_to_spat(shtns_cfg shtns, cplx* d_Qlm, cplx* d_Slm, cplx* d_Tlm, double* d_Vr, double* d_Vt, double* d_Vp, int llim)
{
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	if (llim < mmax*mres)	mmax = llim / mres;	// truncate mmax too !

	cuda_SH_to_spat<0,1>(shtns, d_Qlm, d_Vr, llim, mmax);
	cu_SHsphtor_to_spat(shtns, d_Slm, d_Tlm, d_Vt, d_Vp, llim);
}


extern "C"
void cu_spat_to_SH(shtns_cfg shtns, double *d_Vr, cplx* d_Qlm, int llim)
{
	cuda_spat_to_SH<0,1>(shtns, d_Vr, d_Qlm, llim);
}

extern "C"
void cu_spat_to_SH_float(shtns_cfg shtns, float *d_Vr, cplx_f* d_Qlm, int llim)
{
	cuda_spat_to_SH<0,1,float>(shtns, d_Vr, d_Qlm, llim);
}

extern "C"
void cu_spat_to_SHsphtor(shtns_cfg shtns, double *Vt, double *Vp, cplx *Slm, cplx *Tlm, int llim)
{
	hipError_t err = hipSuccess;
	const long nlm_stride = shtns->nlm_stride;
	double* d_vwlm = shtns->gpu_mem;

	// SHT on the GPU
//	cuda_spat_to_SH<1,1>(shtns, Vt, (cplx*) d_vwlm, llim+1);
//	cuda_spat_to_SH<1,1>(shtns, Vp, (cplx*) (d_vwlm + nlm_stride), llim+1);
	cuda_spat_to_SH<1,2>(shtns, Vt, (cplx*) d_vwlm, llim+1, Vp-Vt);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("spat_to_SHsphtor CUDA error : %s!\n", hipGetErrorString(err));	return; }

	scal2sphtor_gpu(shtns, (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), Slm, Tlm, llim);
}


extern "C"
void cu_spat_to_SHqst(shtns_cfg shtns, double *Vr, double *Vt, double *Vp, cplx *Qlm, cplx *Slm, cplx *Tlm, int llim)
{
	cuda_spat_to_SH<0,1>(shtns, Vr, Qlm, llim);
	cu_spat_to_SHsphtor(shtns, Vt,Vp, Slm,Tlm, llim);
}


/*******************************************************
 * TRANSFORMS OF HOST DATA, INCLUDING TRANSFERS TO GPU *
 *******************************************************/ 

extern "C"
void SH_to_spat_gpu(shtns_cfg shtns, cplx *Qlm, double *Vr, const long int llim)
{
	hipError_t err = hipSuccess;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const int mres = shtns->mres;
	long nlm = shtns->nlm;
	int mmax = shtns->mmax;

	double *d_qlm;
	double *d_q;
	// get pointers to gpu buffers.
	d_qlm = shtns->gpu_mem;
	d_q = d_qlm + shtns->nlm_stride;

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// copy spectral data to GPU
	err = hipMemcpy(d_qlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("SH_to_spat_gpu failed copy qlm\n");	return; }

	// SHT on the GPU
	cuda_SH_to_spat<0,1>(shtns, (cplx*) d_qlm, d_q, llim, mmax);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat_gpu CUDA error : %s!\n", hipGetErrorString(err));	return; }

	// copy back spatial data
	err = hipMemcpy(Vr, d_q, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { printf("SH_to_spat_gpu failed copy back: %s\n", hipGetErrorString(err));	return; }
}

extern "C"
void SH_to_spat_gpu_hostfft(shtns_cfg shtns, cplx *Qlm, double *Vr, const long int llim)
{
	hipError_t err = hipSuccess;
	double *d_qlm;
	double *d_q;
	const long nlat = shtns->nlat;
	const long nphi = shtns->nphi;
	long nlm = shtns->nlm;

	// get pointers to gpu buffers.
	d_qlm = shtns->gpu_mem;
	d_q = d_qlm + shtns->nlm_stride;
	
	double* VrF = Vr;
	if (shtns->fftc_mode > 0)	VrF = shtns->xfft_cpu;

	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}

	// copy spectral data to GPU
	err = hipMemcpy(d_qlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("SH_to_spat_gpu_hostfft failed copy H2D; %s\n", hipGetErrorString(err));	return; }

	// Legendre transform on gpu
	legendre<0,1>(shtns, d_qlm, d_q, llim, mmax);

	// copy back spatial data (before FFT)
	if (12*(mmax+1) < 5*nphi) {
		// copy in two parts, to avoid copying zeros:
		memzero_omp(VrF + nlat*(mmax+1), nlat*(nphi-(2*mmax+1)));
		err = hipMemcpy(VrF, d_q, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost);
		err = hipMemcpy(VrF + nlat*(nphi-mmax), d_q + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost);
	} else {
		err = hipMemcpy(VrF, d_q, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
	}
	if (err != hipSuccess) { printf("SH_to_spat_gpu_hostfft failed copy D2H: %s\n", hipGetErrorString(err));	return; }

	fourier_to_spat_host(shtns, VrF, Vr);
}

extern "C"
void spat_to_SH_gpu_hostfft(shtns_cfg shtns, double *Vr, cplx *Qlm, const long int llim)
{
	hipError_t err = hipSuccess;
	long nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const long nlat = shtns->nlat;
	const long nphi = shtns->nphi;

	// get pointers to gpu buffers.
	double* d_qlm = shtns->gpu_mem;
	double* d_q = d_qlm + shtns->nlm_stride;

	double *VrF = Vr;
	if (shtns->fftc_mode > 0)	VrF = shtns->xfft_cpu;
	spat_to_fourier_host(shtns, Vr, VrF);

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// copy FFT data to GPU
	if (12*(mmax+1) < 5*nphi) {
		// copy in two parts, to avoid copying useless data:
		err = hipMemcpy(d_q, VrF, nlat*(mmax+1)*sizeof(double), hipMemcpyHostToDevice);
		err = hipMemcpy(d_q + nlat*(nphi-mmax), VrF + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyHostToDevice);
	} else {
		err = hipMemcpy(d_q, VrF, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	}
	if (err != hipSuccess) { printf("spat_to_SH_gpu_hostfft failed copy H2D: %s\n", hipGetErrorString(err));	return; }

	// Legendre transform on gpu
	ilegendre<0,1>(shtns, d_q, d_qlm, llim);

	if (nlm < shtns->nlm)	memset(Qlm + nlm, 0, (shtns->nlm - nlm)*2*sizeof(double));
	// copy back spectral data from GPU
	err = hipMemcpy(Qlm, d_qlm, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { printf("spat_to_SH_gpu_hostfft failed copy D2H: %s\n", hipGetErrorString(err));	return; }
}


/** \internal convert from vector SH to scalar SH
	Vlm =  st*d(Slm)/dtheta + I*m*Tlm
	Wlm = -st*d(Tlm)/dtheta + I*m*Slm
**/
/*void sphtor2scal(shtns_cfg shtns, cplx* Slm, cplx* Tlm, cplx* Vlm, cplx* Wlm, const int llim)
{
	const int mmax = shtns->mmax;
	const int lmax = shtns->lmax;
	const int mres = shtns->mres;
	for (int im=0; im<=mmax; im++) {
	const int m = im*mres;
	long l = (im*(2*(lmax+1)-(m+mres)))>>1;
	double* mx = shtns->mx_stdt + 2*l;
	cplx* Sl = (cplx*) &Slm[l];	// virtual pointer for l=0 and im
	cplx* Tl = (cplx*) &Tlm[l];
	cplx* Vl = (cplx*) &Vlm[l+im];
	cplx* Wl = (cplx*) &Wlm[l+im];
	const double em = m;

	cplx sl = Sl[m];
	cplx tl = Tl[m];
	cplx vs = 0.0;
	cplx wt = 0.0;
	for (int l=m; l<=llim; l++) {
		double mxu = mx[2*l];
		double mxl = mx[2*l+1];	// mxl for next l
		vs += I*em*tl;
		wt += I*em*sl;
		cplx vs1 = mxl*sl;		// vs for next l
		cplx wt1 = -mxl*tl;		// wt for next l
		if (l<llim) {
		sl = Sl[l+1];		// kept for next l
		tl = Tl[l+1];
		vs += mxu*sl;
		wt -= mxu*tl;
		}
		Vl[l] = vs;
		Wl[l] = wt;
		vs = vs1;		wt = wt1;
	}
	Vl[llim+1] = vs;
	Wl[llim+1] = wt;
	}
}
*/

extern "C"
void SHsphtor_to_spat_gpu(shtns_cfg shtns, cplx *Slm, cplx *Tlm, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;

	double* d_vwlm = shtns->gpu_mem;
	double* d_vtp = d_vwlm + 2*nlm_stride;

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// transfer and convert on gpu
	err = hipMemcpy(d_vtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpy(d_vtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	sphtor2scal_gpu(shtns, (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	cuda_SH_to_spat<1,1>(shtns, (cplx*) d_vwlm, d_vtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht, hipEventDisableTiming );
	hipEventRecord(ev_sht, shtns->comp_stream);					// record the end of scalar SH (theta).

	cuda_SH_to_spat<1,1>(shtns, (cplx*) (d_vwlm + nlm_stride), d_vtp + spat_stride, llim+1, mmax);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(Vt, d_vtp, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipEventDestroy(ev_sht);

	// copy back spatial data (phi)
	err = hipMemcpy(Vp, d_vtp + spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
}

extern "C"
void SHsphtor_to_spat_gpu_hostfft(shtns_cfg shtns, cplx *Slm, cplx *Tlm, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht, ev_sht2;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;

	double* d_vwlm = shtns->gpu_mem;
	double* d_vtp = d_vwlm + 2*nlm_stride;
	double* VtF = Vt;
	double* VpF = Vp;
	if (shtns->fftc_mode > 0) {
		VtF = Vp;
		VpF = shtns->xfft_cpu;
	}

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// transfer and convert on gpu
	err = hipMemcpy(d_vtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpy(d_vtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	sphtor2scal_gpu(shtns, (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	legendre<1,1>(shtns, d_vwlm, d_vtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht, hipEventDisableTiming );
	hipEventRecord(ev_sht, shtns->comp_stream);					// record the end of scalar SH (theta).

	legendre<1,1>(shtns, (d_vwlm + nlm_stride), d_vtp + spat_stride, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht2, hipEventDisableTiming );
	hipEventRecord(ev_sht2, shtns->comp_stream);					// record the end of scalar SH (phi).

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(VtF, d_vtp, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipMemcpyAsync(VtF + nlat*(nphi-mmax), d_vtp + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipEventRecord(ev_sht, shtns->xfer_stream);

	hipStreamWaitEvent(xfer_stream, ev_sht2, 0);					// xfer stream waits for end of scalar SH (phi).
	hipMemcpyAsync(VpF, d_vtp + spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipMemcpyAsync(VpF + nlat*(nphi-mmax), d_vtp + spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipEventRecord(ev_sht2, shtns->xfer_stream);

	memzero_omp(Vt + nlat*(mmax+1), Vp + nlat*(mmax+1), nlat*(nphi-(2*mmax+1)));

	hipEventSynchronize(ev_sht);
	fourier_to_spat_host(shtns, VtF, Vt);
	hipEventSynchronize(ev_sht2);
	fourier_to_spat_host(shtns, VpF, Vp);

	hipEventDestroy(ev_sht2);
	hipEventDestroy(ev_sht);
}

extern "C"
void SHsphtor_to_spat_gpu2_hostfft(shtns_cfg shtns, cplx *Slm, cplx *Tlm, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht, ev_sht2;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;

	double* d_vwlm = shtns->gpu_mem;
	double* d_vtp = d_vwlm + 2*nlm_stride;
	double* VtF = Vt;
	double* VpF = Vp;
	if (shtns->fftc_mode > 0) {
		VtF = Vp;
		VpF = shtns->xfft_cpu;
	}

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// transfer and convert on gpu
	err = hipMemcpy(d_vtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpy(d_vtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	sphtor2scal_gpu(shtns, (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	legendre<1,2>(shtns, d_vwlm, d_vtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht, hipEventDisableTiming );
	hipEventRecord(ev_sht, shtns->comp_stream);					// record the end of scalar SH (theta+phi).

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(VtF, d_vtp, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipMemcpyAsync(VtF + nlat*(nphi-mmax), d_vtp + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipEventRecord(ev_sht, shtns->xfer_stream);

	hipMemcpyAsync(VpF, d_vtp + spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipMemcpyAsync(VpF + nlat*(nphi-mmax), d_vtp + spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, shtns->xfer_stream);
	hipEventCreateWithFlags(&ev_sht2, hipEventDisableTiming );
	hipEventRecord(ev_sht2, shtns->xfer_stream);

	memzero_omp(Vt + nlat*(mmax+1), Vp + nlat*(mmax+1), nlat*(nphi-(2*mmax+1)));

	hipEventSynchronize(ev_sht);
	fourier_to_spat_host(shtns, VtF, Vt);
	hipEventSynchronize(ev_sht2);
	fourier_to_spat_host(shtns, VpF, Vp);

	hipEventDestroy(ev_sht2);
	hipEventDestroy(ev_sht);
}

extern "C"
void SHsphtor_to_spat_gpu2(shtns_cfg shtns, cplx *Slm, cplx *Tlm, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;

	double* d_vwlm;
	double* d_vtp;

	d_vwlm = shtns->gpu_mem;
	d_vtp = d_vwlm + 2*nlm_stride;

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	// transfer and convert on gpu
	err = hipMemcpy(d_vtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpy(d_vtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	sphtor2scal_gpu(shtns, (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	cuda_SH_to_spat<1,2>(shtns, (cplx*) d_vwlm, d_vtp, llim+1, mmax);		// Vt and Vp together  (merge with sphtor2scal_gpu)
	hipMemcpy(Vt, d_vtp, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Vp, d_vtp + spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
}


extern "C"
void SHqst_to_spat_gpu(shtns_cfg shtns, cplx *Qlm, cplx *Slm, cplx *Tlm, double *Vr, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht0, ev_sht1, ev_up;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm = shtns->gpu_mem;
	double* d_vrtp = d_qvwlm + 2*nlm_stride;

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	/// 1) start scalar SH for radial component.
	err = hipMemcpy(d_qvwlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 0 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	cuda_SH_to_spat<0,1>(shtns, (cplx*) d_qvwlm, d_vrtp + 2*spat_stride, llim, mmax);

	// OR transfer and convert on gpu
	err = hipMemcpyAsync(d_vrtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpyAsync(d_vrtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	hipEventCreateWithFlags(&ev_sht0, hipEventDisableTiming );
	hipEventRecord(ev_sht0, comp_stream);					// record the end of scalar SH (radial).
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);			// record the end of upload
	hipStreamWaitEvent(comp_stream, ev_up, 0);				// compute stream waits for end of transfer.

	sphtor2scal_gpu(shtns, (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	cuda_SH_to_spat<1,1>(shtns, (cplx*) d_qvwlm, d_vrtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht1, hipEventDisableTiming );
	hipEventRecord(ev_sht1, comp_stream);					// record the end of scalar SH (theta).

	cuda_SH_to_spat<1,1>(shtns, (cplx*) (d_qvwlm + nlm_stride), d_vrtp + spat_stride, llim+1, mmax);

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht0, 0);					// xfer stream waits for end of scalar SH (radial).
	hipMemcpyAsync(Vr, d_vrtp + 2*spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventDestroy(ev_sht0);

	hipStreamWaitEvent(xfer_stream, ev_sht1, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(Vt, d_vrtp, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventDestroy(ev_sht1);

	// copy back the last transform (compute stream).
	err = hipMemcpy(Vp, d_vrtp + spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);

	hipEventDestroy(ev_up);
}

extern "C"
void SHqst_to_spat_gpu_hostfft(shtns_cfg shtns, cplx *Qlm, cplx *Slm, cplx *Tlm, double *Vr, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht0, ev_sht1, ev_sht2, ev_up;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm = shtns->gpu_mem;
	double* d_vrtp = d_qvwlm + 2*nlm_stride;
	
	double* VrF = Vr;
	double* VtF = Vt;
	double* VpF = Vp;
	if (shtns->fftc_mode > 0) {
		VrF = Vt;
		VtF = Vp;
		VpF = shtns->xfft_cpu;
	}

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	/// 1) start scalar SH for radial component.
	err = hipMemcpy(d_qvwlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 0 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	legendre<0,1>(shtns, d_qvwlm, d_vrtp + 2*spat_stride, llim, mmax);

	// OR transfer and convert on gpu
	err = hipMemcpyAsync(d_vrtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpyAsync(d_vrtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	hipEventCreateWithFlags(&ev_sht0, hipEventDisableTiming );
	hipEventRecord(ev_sht0, comp_stream);					// record the end of scalar SH (radial).
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);			// record the end of upload
	hipStreamWaitEvent(comp_stream, ev_up, 0);				// compute stream waits for end of transfer.

	sphtor2scal_gpu(shtns, (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	legendre<1,1>(shtns, d_qvwlm, d_vrtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht1, hipEventDisableTiming );
	hipEventRecord(ev_sht1, comp_stream);					// record the end of scalar SH (theta).

	legendre<1,1>(shtns, (d_qvwlm + nlm_stride), d_vrtp + spat_stride, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht2, hipEventDisableTiming );
	hipEventRecord(ev_sht2, comp_stream);					// record the end of scalar SH (phi).

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht0, 0);					// xfer stream waits for end of scalar SH (radial).
	hipMemcpyAsync(VrF, d_vrtp + 2*spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VrF + nlat*(nphi-mmax), d_vrtp + 2*spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventRecord(ev_sht0, xfer_stream);

	hipStreamWaitEvent(xfer_stream, ev_sht1, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(VtF, d_vrtp, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VtF + nlat*(nphi-mmax), d_vrtp + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventRecord(ev_sht1, xfer_stream);

	hipStreamWaitEvent(xfer_stream, ev_sht2, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(VpF, d_vrtp + spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VpF + nlat*(nphi-mmax), d_vrtp + spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventRecord(ev_sht2, xfer_stream);

	memzero_omp(Vr + nlat*(mmax+1), Vt + nlat*(mmax+1), Vp + nlat*(mmax+1), nlat*(nphi-(2*mmax+1)));

	hipEventSynchronize(ev_sht0);
	fourier_to_spat_host(shtns, VrF, Vr);
	hipEventSynchronize(ev_sht1);
	fourier_to_spat_host(shtns, VtF, Vt);
	hipEventSynchronize(ev_sht2);
	fourier_to_spat_host(shtns, VpF, Vp);

	hipEventDestroy(ev_sht0);
	hipEventDestroy(ev_sht1);
	hipEventDestroy(ev_up);
}

extern "C"
void SHqst_to_spat_gpu2_hostfft(shtns_cfg shtns, cplx *Qlm, cplx *Slm, cplx *Tlm, double *Vr, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht0, ev_sht1, ev_sht2, ev_up;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm = shtns->gpu_mem;
	double* d_vrtp = d_qvwlm + 2*nlm_stride;
	
	double* VrF = Vr;
	double* VtF = Vt;
	double* VpF = Vp;
	if (shtns->fftc_mode > 0) {
		VrF = Vt;
		VtF = Vp;
		VpF = shtns->xfft_cpu;
	}

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	/// 1) start scalar SH for radial component.
	err = hipMemcpy(d_qvwlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 0 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	legendre<0,1>(shtns, d_qvwlm, d_vrtp + 2*spat_stride, llim, mmax);

	// OR transfer and convert on gpu
	err = hipMemcpyAsync(d_vrtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpyAsync(d_vrtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	hipEventCreateWithFlags(&ev_sht0, hipEventDisableTiming );
	hipEventRecord(ev_sht0, comp_stream);					// record the end of scalar SH (radial).
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);			// record the end of upload
	hipStreamWaitEvent(comp_stream, ev_up, 0);				// compute stream waits for end of transfer.

	sphtor2scal_gpu(shtns, (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	legendre<1,2>(shtns, d_qvwlm, d_vrtp, llim+1, mmax);
	hipEventCreateWithFlags(&ev_sht1, hipEventDisableTiming );
	hipEventRecord(ev_sht1, comp_stream);					// record the end of scalar SH (theta+phi).

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht0, 0);					// xfer stream waits for end of scalar SH (radial).
	hipMemcpyAsync(VrF, d_vrtp + 2*spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VrF + nlat*(nphi-mmax), d_vrtp + 2*spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventRecord(ev_sht0, xfer_stream);

	hipStreamWaitEvent(xfer_stream, ev_sht1, 0);					// xfer stream waits for end of scalar SH (theta).
	hipMemcpyAsync(VtF, d_vrtp, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VtF + nlat*(nphi-mmax), d_vrtp + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventRecord(ev_sht1, xfer_stream);

	hipMemcpyAsync(VpF, d_vrtp + spat_stride, nlat*(mmax+1)*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipMemcpyAsync(VpF + nlat*(nphi-mmax), d_vrtp + spat_stride + nlat*(nphi-mmax), nlat*mmax*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventCreateWithFlags(&ev_sht2, hipEventDisableTiming );
	hipEventRecord(ev_sht2, xfer_stream);

	memzero_omp(Vr + nlat*(mmax+1), Vt + nlat*(mmax+1), Vp + nlat*(mmax+1), nlat*(nphi-(2*mmax+1)));

	hipEventSynchronize(ev_sht0);
	fourier_to_spat_host(shtns, VrF, Vr);
	hipEventSynchronize(ev_sht1);
	fourier_to_spat_host(shtns, VtF, Vt);
	hipEventSynchronize(ev_sht2);
	fourier_to_spat_host(shtns, VpF, Vp);

	hipEventDestroy(ev_sht2);
	hipEventDestroy(ev_sht1);
	hipEventDestroy(ev_sht0);
	hipEventDestroy(ev_up);
}


extern "C"
void SHqst_to_spat_gpu2(shtns_cfg shtns, cplx *Qlm, cplx *Slm, cplx *Tlm, double *Vr, double *Vt, double *Vp, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_sht0, ev_up;
	int nlm = shtns->nlm;
	int mmax = shtns->mmax;
	const int mres = shtns->mres;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm = shtns->gpu_mem;
	double* d_vrtp = d_qvwlm + 2*nlm_stride;

	if (llim < mmax*mres) {
		mmax = llim / mres;	// truncate mmax too !
		nlm = nlm_calc( shtns->lmax, mmax, mres);		// transfer less data
	}
	/// 1) start scalar SH for radial component.
	err = hipMemcpy(d_qvwlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 0 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	cuda_SH_to_spat<0,1>(shtns, (cplx*) d_qvwlm, d_vrtp + 2*spat_stride, llim, mmax);

	// OR transfer and convert on gpu
	err = hipMemcpyAsync(d_vrtp, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpyAsync(d_vrtp + nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }

	hipEventCreateWithFlags(&ev_sht0, hipEventDisableTiming );
	hipEventRecord(ev_sht0, comp_stream);					// record the end of scalar SH (radial).
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);			// record the end of upload
	hipStreamWaitEvent(comp_stream, ev_up, 0);				// compute stream waits for end of transfer.

	sphtor2scal_gpu(shtns, (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), llim, mmax);

	// SHT on the GPU
	cuda_SH_to_spat<1,2>(shtns, (cplx*) d_qvwlm, d_vrtp, llim+1, mmax);

	err = hipGetLastError();
	if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

	hipStreamWaitEvent(xfer_stream, ev_sht0, 0);					// xfer stream waits for end of scalar SH (radial).
	hipMemcpyAsync(Vr, d_vrtp + 2*spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	hipEventDestroy(ev_sht0);

	hipMemcpy(Vt, d_vrtp, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);

	// copy back the last transform (compute stream).
	err = hipMemcpy(Vp, d_vrtp + spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);

	hipEventDestroy(ev_up);
}


extern "C"
void spat_to_SH_gpu(shtns_cfg shtns, double *Vr, cplx *Qlm, const long int llim)
{
	hipError_t err = hipSuccess;
	const int nlm = shtns->nlm;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;

	double *d_qlm;
	double *d_q;

	// Allocate the device work vectors qlm and q
    //err = hipMalloc((void **)&d_qlm, ((2*nlm +31 + nlat*nphi+31)/32)*32*sizeof(double));
    d_qlm = shtns->gpu_mem;
    d_q = d_qlm + shtns->nlm_stride;

	// copy spatial data to GPU
	err = hipMemcpy(d_q, Vr, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("spat_to_SH_gpu failed copy q\n");	return; }

	// SHT on the GPU
	cu_spat_to_SH(shtns, d_q, (cplx*) d_qlm, llim);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("spat_to_SH_gpu CUDA error : %s!\n", hipGetErrorString(err));	return; }

	// copy back spectral data
	err = hipMemcpy(Qlm, d_qlm, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { printf("spat_to_SH_gpu failed copy back\n");	return; }

	//hipFree(d_qlm);
}


extern "C"
void spat_to_SHsphtor_gpu(shtns_cfg shtns, double *Vt, double *Vp, cplx *Slm, cplx *Tlm, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_up;
	const int nlm = shtns->nlm;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;

	double* d_vwlm;
	double* d_vtp;

	//err = hipMalloc( (void **)&d_vwlm, (4*nlm_stride + 2*spat_stride)*sizeof(double) );
	d_vtp = shtns->gpu_mem;
	d_vwlm = d_vtp + 2*spat_stride;

	// copy spatial data to gpu
	err = hipMemcpy(d_vtp, Vt, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 3 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	cuda_spat_to_SH<1,1>(shtns, d_vtp, (cplx*) d_vwlm, llim+1);

	err = hipMemcpyAsync(d_vtp + spat_stride, Vp, nlat*nphi*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 4 error : %s!\n", hipGetErrorString(err));	return; }
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);				// record the end of scalar SH (theta).
	hipStreamWaitEvent(shtns->comp_stream, ev_up, 0);					// compute stream waits for end of data transfer (phi).
	cuda_spat_to_SH<1,1>(shtns, d_vtp + spat_stride, (cplx*) (d_vwlm + nlm_stride), llim+1);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("spat_to_SHsphtor CUDA error : %s!\n", hipGetErrorString(err));	return; }

	scal2sphtor_gpu(shtns, (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), llim);

	err = hipMemcpy(Slm, d_vtp, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
	err = hipMemcpy(Tlm, d_vtp+nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);

	hipEventDestroy(ev_up);
//    hipFree(d_vwlm);
//    hipHostFree(vw);
}

extern "C"
void spat_to_SHsphtor_gpu2(shtns_cfg shtns, double *Vt, double *Vp, cplx *Slm, cplx *Tlm, const long int llim)
{
	hipError_t err = hipSuccess;
	const int nlm = shtns->nlm;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;

	double* d_vwlm;
	double* d_vtp;

	//err = hipMalloc( (void **)&d_vwlm, (4*nlm_stride + 2*spat_stride)*sizeof(double) );
	d_vtp = shtns->gpu_mem;
	d_vwlm = d_vtp + 2*spat_stride;

	// copy spatial data to gpu
	err = hipMemcpy(d_vtp, Vt, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	err = hipMemcpy(d_vtp + spat_stride, Vp, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 3 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	cuda_spat_to_SH<1,2>(shtns, d_vtp, (cplx*) d_vwlm, llim+1);

	scal2sphtor_gpu(shtns, (cplx*) d_vwlm, (cplx*) (d_vwlm+nlm_stride), (cplx*) d_vtp, (cplx*) (d_vtp+nlm_stride), llim);

	err = hipMemcpy(Slm, d_vtp, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
	err = hipMemcpy(Tlm, d_vtp+nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
}


extern "C"
void spat_to_SHqst_gpu(shtns_cfg shtns, double *Vr, double *Vt, double *Vp, cplx *Qlm, cplx *Slm, cplx *Tlm, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_up, ev_up2, ev_sh2;
	const int nlm = shtns->nlm;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm;
	double* d_vrtp;

	// Allocate the device work vectors
//	err = hipMalloc( (void **)&d_qvwlm, (5*nlm_stride + 3*spat_stride)*sizeof(double) );
	d_qvwlm = shtns->gpu_mem;
	d_vrtp = d_qvwlm + 2*nlm_stride;

	// copy spatial data to gpu
	err = hipMemcpy(d_vrtp, Vt, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 3 error : %s!\n", hipGetErrorString(err));	return; }
	// SHT on the GPU
	cuda_spat_to_SH<1,1>(shtns, d_vrtp, (cplx*) d_qvwlm, llim+1);

	err = hipMemcpyAsync(d_vrtp + spat_stride, Vp, nlat*nphi*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 4 error : %s!\n", hipGetErrorString(err));	return; }
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);				// record the end of scalar SH (theta).
	hipStreamWaitEvent(comp_stream, ev_up, 0);			// compute stream waits for end of data transfer (phi).
	cuda_spat_to_SH<1,1>(shtns, d_vrtp + spat_stride, (cplx*) (d_qvwlm + nlm_stride), llim+1);
	err = hipGetLastError();
	if (err != hipSuccess) { printf("spat_to_SHsphtor CUDA error : %s!\n", hipGetErrorString(err));	return; }

	scal2sphtor_gpu(shtns, (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), llim);
	hipEventCreateWithFlags(&ev_sh2, hipEventDisableTiming );
	hipEventRecord(ev_sh2, comp_stream);				// record the end of vector transform.

	err = hipMemcpyAsync(d_vrtp + 2*spat_stride, Vr, nlat*nphi*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 5 error : %s!\n", hipGetErrorString(err));	return; }
	hipEventCreateWithFlags(&ev_up2, hipEventDisableTiming );
	hipEventRecord(ev_up2, xfer_stream);				// record the end of scalar SH (theta).
	hipStreamWaitEvent(comp_stream, ev_up2, 0);		// compute stream waits for end of data transfer (phi).
	// scalar SHT on the GPU
	cuda_spat_to_SH<0,1>(shtns, d_vrtp + 2*spat_stride, (cplx*) d_qvwlm, llim);

	hipStreamWaitEvent(xfer_stream, ev_sh2, 0);					// xfer stream waits for end of vector sht.
	err = hipMemcpyAsync(Slm, d_vrtp, 2*nlm*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	err = hipMemcpyAsync(Tlm, d_vrtp+nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);

	err = hipMemcpy(Qlm, d_qvwlm, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);

	hipEventDestroy(ev_up);	hipEventDestroy(ev_up2);	hipEventDestroy(ev_sh2);
//    hipFree(d_qvwlm);
//    hipHostFree(vw);
}


extern "C"
void spat_to_SHqst_gpu2(shtns_cfg shtns, double *Vr, double *Vt, double *Vp, cplx *Qlm, cplx *Slm, cplx *Tlm, const long int llim)
{
	hipError_t err = hipSuccess;
	hipEvent_t ev_up, ev_sh2;
	const int nlm = shtns->nlm;
	const int nlat = shtns->nlat;
	const int nphi = shtns->nphi;
	const long nlm_stride = shtns->nlm_stride;
	const long spat_stride = shtns->spat_stride;
	hipStream_t xfer_stream = shtns->xfer_stream;
	hipStream_t comp_stream = shtns->comp_stream;

	double* d_qvwlm;
	double* d_vrtp;

	d_qvwlm = shtns->gpu_mem;
	d_vrtp = d_qvwlm + 2*nlm_stride;

	// copy Vt and Vp to gpu (async)
	err = hipMemcpy(d_vrtp, Vt, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 4 error : %s!\n", hipGetErrorString(err));	return; }
	err = hipMemcpy(d_vrtp + spat_stride, Vp, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) { printf("memcpy 3 error : %s!\n", hipGetErrorString(err));	return; }
	cuda_spat_to_SH<1,2>(shtns, d_vrtp, (cplx*) d_qvwlm, llim+1);
	scal2sphtor_gpu(shtns, (cplx*) d_qvwlm, (cplx*) (d_qvwlm+nlm_stride), (cplx*) d_vrtp, (cplx*) (d_vrtp+nlm_stride), llim);
	hipEventCreateWithFlags(&ev_sh2, hipEventDisableTiming );
	hipEventRecord(ev_sh2, comp_stream);				// record the end of vector transform.

	// copy Vr to gpu
	err = hipMemcpyAsync(d_vrtp + 2*spat_stride, Vr, nlat*nphi*sizeof(double), hipMemcpyHostToDevice, xfer_stream);
	if (err != hipSuccess) { printf("memcpy 5 error : %s!\n", hipGetErrorString(err));	return; }
	hipEventCreateWithFlags(&ev_up, hipEventDisableTiming );
	hipEventRecord(ev_up, xfer_stream);				// record the end of data transfer.
	hipStreamWaitEvent(comp_stream, ev_up, 0);			// compute stream waits for end of data transfer.
	// scalar SHT on the GPU
	cuda_spat_to_SH<0,1>(shtns, d_vrtp + 2*spat_stride, (cplx*) d_qvwlm, llim);

	// copy back
	hipStreamWaitEvent(xfer_stream, ev_sh2, 0);					// xfer stream waits for end of vector sht.
	err = hipMemcpyAsync(Slm, d_vrtp, 2*nlm*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);
	err = hipMemcpyAsync(Tlm, d_vrtp+nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost, xfer_stream);

	err = hipMemcpy(Qlm, d_qvwlm, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);

	hipEventDestroy(ev_up);	hipEventDestroy(ev_sh2);
}

void* fgpu[4][SHT_NTYP] = {
	{ (void*) SH_to_spat_gpu, (void*) spat_to_SH_gpu, (void*) SHsphtor_to_spat_gpu, (void*) spat_to_SHsphtor_gpu, 0, 0, (void*) SHqst_to_spat_gpu, (void*) spat_to_SHqst_gpu },
	{ 0, 0, (void*) SHsphtor_to_spat_gpu2, (void*) spat_to_SHsphtor_gpu2, 0, 0, (void*) SHqst_to_spat_gpu2, (void*) spat_to_SHqst_gpu2 },
	{ (void*) SH_to_spat_gpu_hostfft, (void*) spat_to_SH_gpu_hostfft, (void*) SHsphtor_to_spat_gpu_hostfft, 0, 0, 0, (void*) SHqst_to_spat_gpu2_hostfft, 0 },
	{ 0, 0, (void*) SHsphtor_to_spat_gpu2_hostfft, 0, 0, 0, (void*) SHqst_to_spat_gpu2_hostfft, 0}
};
