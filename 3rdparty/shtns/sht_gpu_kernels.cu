#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2021 Centre National de la Recherche Scientifique.
 * written by Nathanael Schaeffer (CNRS, ISTerre, Grenoble, France).
 * 
 * nathanael.schaeffer@univ-grenoble-alpes.fr
 * 
 * This software is governed by the CeCILL license under French law and
 * abiding by the rules of distribution of free software. You can use,
 * modify and/or redistribute the software under the terms of the CeCILL
 * license as circulated by CEA, CNRS and INRIA at the following URL
 * "http://www.cecill.info".
 * 
 * The fact that you are presently reading this means that you have had
 * knowledge of the CeCILL license and that you accept its terms.
 * 
 */

// Various CUDA kernels for SHTns

// adjustment for cuda
#undef SHT_L_RESCALE_FLY
#undef SHT_ACCURACY
#define SHT_L_RESCALE_FLY 1800
#define SHT_L_RESCALE_FLY_FLOAT 128
#define SHT_ACCURACY 1.0e-40
#define SHT_ACCURACY_FLOAT 1.0e-15
#define SHT_SCALE_FACTOR_FLOAT 72057594037927936.0


#if (__CUDACC_VER_MAJOR__ < 8) || ( defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600 )
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
							 (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
	old = atomicCAS(address_as_ull, assumed,
						__double_as_longlong(val +
							   __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}
#endif

// define our own suffle macros, to accomodate cuda<9 and cuda>=9
#if __CUDACC_VER_MAJOR__ < 9
	#define shfl_xor(...) __shfl_xor(__VA_ARGS__)
	#define shfl_down(...) __shfl_down(__VA_ARGS__)
	#define shfl(...) __shfl(__VA_ARGS__)
	#define _any(p) __any(p)
	#define _all(p) __all(p)
#else
	#define shfl_xor(...) __shfl_xor_sync(0xFFFFFFFF, __VA_ARGS__)
	#define shfl_down(...) __shfl_down_sync(0xFFFFFFFF, __VA_ARGS__)
	#define shfl(...) __shfl_sync(0xFFFFFFFF, __VA_ARGS__)
	#define _any(p) __any_sync(0xFFFFFFFF, p)
	#define _all(p) __all_sync(0xFFFFFFFF, p)
#endif

/*
__device__ __forceinline__ int getLaneId() {
  int laneId;
  asm("mov.s32 %0, %laneid;" : "=r"(laneId) );
  return laneId;
}

__device__ __forceinline__ void namedBarrierWait(int name, int numThreads) {
  asm volatile("bar.sync %0, %1;" : : "r"(name), "r"(numThreads) : "memory");
}

__device__ __forceinline__ void namedBarrierArrived(int name, int numThreads) {
  asm volatile("bar.arrive %0, %1;" : : "r"(name), "r"(numThreads) : "memory");
}
*/

__global__ void copy_kernel(const double *in, float *out, const int n) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
		out[i] = in[i];
}

static void copy_convert(shtns_cfg shtns) {
	const int BLOCKSIZE = 256;		// good value
	int n = 2*shtns->nlm;
	copy_kernel<<<(n+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE, 0, shtns->comp_stream>>>(shtns->d_alm, shtns->d_alm_f, n);
	n = 2*shtns->nlat_2;
	copy_kernel<<<(n+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE, 0, shtns->comp_stream>>>(shtns->d_ct, shtns->d_ct_f, n);
}


/// dim0, dim1 : size in complex numbers !
/// BLOCK_DIM_Y must be between 1 and 16
template<int BLOCK_DIM_Y, typename real> __global__ void
transpose_cplx_kernel(const real* in, real* out, const int dim0, const int dim1)
{
	const int TILE_DIM = WARPSZE/2;		// 16 complex numbers per warp, read as 32 reals.
	__shared__ real shrdMem[TILE_DIM][TILE_DIM+1][2];		// avoid shared mem conflicts

	const int lx = threadIdx.x >> 1;
	const int ly = threadIdx.y;
	const int ri = threadIdx.x & 1;		// real/imag index

	const int bx = TILE_DIM * blockIdx.x;
	const int by = TILE_DIM * blockIdx.y;

	int gx = lx + bx;
	int gy = ly + by;
	#pragma unroll
	for (int repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
		int gy_ = gy+repeat;
		shrdMem[ly + repeat][lx][ri] = in[2*(gy_ * dim0 + gx) + ri];
	}

	// transpose tiles:
	gx = lx + by;
	gy = ly + bx;

	__syncthreads();
	// transpose within tile:
	#pragma unroll
	for (unsigned repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
		int gy_ = gy+repeat;
		out[2*(gy_ * dim1 + gx) + ri] = shrdMem[lx][ly + repeat][ri];
	}
}

/// dim0, dim1 : size in complex numbers !
/// BLOCK_DIM_Y must be a power of 2 between 1 and 16
template<int BLOCK_DIM_Y, typename real> __global__ void
transpose_cplx_zero_kernel(const real* in, real* out, const int dim0, const int dim1, const int mmax)
{
	const int TILE_DIM = WARPSZE/2;		// 16 complex numbers per warp, read as 32 reals.
	__shared__ real shrdMem[TILE_DIM][TILE_DIM+1][2];		// avoid shared mem conflicts

	const int ly = threadIdx.y;
	const int lx = threadIdx.x >> 1;
	const int ri = threadIdx.x & 1;		// real/imag index

	const int by = TILE_DIM * blockIdx.y;
	const int bx = TILE_DIM * blockIdx.x;

	int gy = ly + by;
	int gx = lx + bx;

	if ((gy+(TILE_DIM-BLOCK_DIM_Y) <= mmax) || (gy >= dim1 - mmax)) {		// SAFE, no zero to insert
		#pragma unroll
		for (int repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
			int gy_ = gy+repeat;
			shrdMem[ly + repeat][lx][ri] = in[2*(gy_ * dim0 + gx) + ri];
		}
	} else {
		for (int repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
			int gy_ = gy+repeat;
			if ((gy_ <= mmax) || (gy_ >= dim1 - mmax)) {
				shrdMem[ly + repeat][lx][ri] = in[2*(gy_ * dim0 + gx) + ri];
			} else {
				shrdMem[ly + repeat][lx][ri] = 0.0;
			}
		}
	}

	// transpose tiles:
	gy = ly + bx;
	gx = lx + by;

	__syncthreads();
	// transpose within tile:
	#pragma unroll
	for (unsigned repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
		int gy_ = gy+repeat;
		out[2*(gy_ * dim1 + gx) + ri] = shrdMem[lx][ly + repeat][ri];
	}
}

/// dim0, dim1 : size in complex numbers !
/// BLOCK_DIM_Y must be a power of 2 between 1 and 16
template<int BLOCK_DIM_Y, typename real> __global__ void
transpose_cplx_skip_kernel(const real* in, real* out, const int dim0, const int dim1, const int mmax)
{
	const int TILE_DIM = WARPSZE/2;		// 16 complex per warp, read as 32 reals.
	__shared__ real shrdMem[TILE_DIM][TILE_DIM+1][2];		// avoid shared mem conflicts

	const int lx = threadIdx.x >> 1;
	const int ly = threadIdx.y;
	const int ri = threadIdx.x & 1;		// real/imag index

	const int bx = TILE_DIM * blockIdx.x;
	const int by = TILE_DIM * blockIdx.y;

	int gx = lx + bx;
	int gy = ly + by;

	if ((gx <= mmax) || (gx >= dim0 - mmax)) {		// read only data if m<=mmax
		#pragma unroll
		for (int repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
			int gy_ = gy+repeat;
			shrdMem[ly + repeat][lx][ri] = in[2*(gy_ * dim0 + gx) + ri];
		}
	}

	// transpose tiles:
	gy = ly + bx;
	gx = lx + by;

	__syncthreads();
	// transpose within tile:
	if ((gy <= mmax) || (gy+(TILE_DIM-BLOCK_DIM_Y) >= dim0 - mmax)) {		// write all useful data (+a bit more), the rest is ignored anyway
		#pragma unroll
		for (unsigned repeat = 0; repeat < TILE_DIM; repeat += BLOCK_DIM_Y) {
			int gy_ = gy+repeat;
			out[2*(gy_ * dim1 + gx) + ri] = shrdMem[lx][ly + repeat][ri];
		}
	}
}

/// dim0, dim1 must be multiple of 16.
template <typename real> static void
transpose_cplx(hipStream_t stream, const real* in, real* out, const int dim0, const int dim1)
{
	const int block_dim_y = 4;		// good performance with 4 (MUST be power of 2 between 1 and 16)
	dim3 blocks(dim0/16, dim1/16);
	dim3 threads(32, block_dim_y);
	transpose_cplx_kernel<block_dim_y,real> <<<blocks, threads, 0, stream>>>(in, out, dim0, dim1);
}

/// dim0, dim1 must be multiple of 16.
template <typename real> static void
transpose_cplx_zero(hipStream_t stream, const real* in, real* out, const int dim0, const int dim1, const int mmax)
{
	const int block_dim_y = 4;		// good performance with 4 (MUST be power of 2 between 1 and 16)
	dim3 blocks(dim0/16, dim1/16);
	dim3 threads(32, block_dim_y);
	transpose_cplx_zero_kernel<block_dim_y,real> <<<blocks, threads, 0, stream>>>(in, out, dim0, dim1, mmax);
}

/// dim0, dim1 must be multiple of 16.
template <typename real> static void
transpose_cplx_skip(hipStream_t stream, const real* in, real* out, const int dim0, const int dim1, const int mmax)
{
	const int block_dim_y = 4;		// good performance with 4 (MUST be power of 2 between 1 and 16)
	dim3 blocks(dim0/16, dim1/16);
	dim3 threads(32, block_dim_y);
	transpose_cplx_skip_kernel<block_dim_y,real> <<<blocks, threads, 0, stream>>>(in, out, dim0, dim1, mmax);
}



/// On KEPLER, This kernel is fastest with THREADS_PER_BLOCK=256 and NW=1
template<int BLOCKSIZE, int S, int NW, typename real=double> __global__ void
leg_m0_kernel(const real* __restrict__ al, const real* __restrict__ ct, const real* __restrict__ ql, real* q, const int llim, const int nlat_2)
{
	// im = 0
	const int it = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = threadIdx.x;

	__shared__ real ak[BLOCKSIZE];		// size blockDim.x
	__shared__ real qk[BLOCKSIZE/2];	// size blockDim.x / 2

	ak[j] = al[j];
	if ((j <= llim)&&(j<blockDim.x/2)) qk[j] = ql[2*j];
	__syncthreads();

	int l = 0;
	int k = 0;	int kq = 0;
	real cost[NW];
	real y0[NW];    real y1[NW];
	real re[NW];    real ro[NW];

	for (int i=0; i<NW; i++) {
	cost[i] = (it+i<nlat_2) ? ct[it+i] : 0.0;
	y0[i] = ak[0];
	if (S==1) y0[i] *= rsqrt(1.0 - cost[i]*cost[i]);	// for vectors, divide by sin(theta)
	}
	for (int i=0; i<NW; i++) {
	re[i] = y0[i] * qk[0];
	y1[i] = y0[i] * ak[1] * cost[i];
	}
	for (int i=0; i<NW; i++) {
	ro[i] = y1[i] * qk[1];
	}
	al+=2;    l+=2;	k+=2;	kq+=2;
	while(l<llim) {
	if (k+6 >= blockDim.x) {
		__syncthreads();
		ak[j] = al[j];
		if ((j <= llim)&&(j<blockDim.x/2)) qk[j] = ql[2*(l+j)];
		k=0;	kq=0;
		__syncthreads();
	}
	for (int i=0; i<NW; i++)	y0[i]  = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
	for (int i=0; i<NW; i++)	re[i] += y0[i] * qk[kq];
	for (int i=0; i<NW; i++)	y1[i]  = ak[k+3]*cost[i]*y0[i] + ak[k+2]*y1[i];
	for (int i=0; i<NW; i++)	ro[i] += y1[i] * qk[kq+1];
	al+=4;	l+=2;	k+=4;	kq+=2;
	}
	if (l==llim) {
	for (int i=0; i<NW; i++)	y0[i]  = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
	for (int i=0; i<NW; i++)	re[i] += y0[i] * qk[kq];
	}

	for (int i=0; i<NW; i++) {
	if (it+i < nlat_2) {
		q[it+i] = re[i]+ro[i];
		q[nlat_2*2-1-(it+i)] = re[i]-ro[i];
	}
	}
/*
	if (it < nlat_2) {
		int l = 0;
		real cost = ct[it];
		real y0 = al[0];
		real re = y0 * ql[0];
		real y1 = y0 * al[1] * cost;
		real ro = y1 * ql[1];
		al+=2;    l+=2;
		while(l<llim) {
			y0  = al[1]*(cost*y1) + al[0]*y0;
			re += y0 * ql[l];
			y1  = al[3]*(cost*y0) + al[2]*y1;
			ro += y1 * ql[l+1];
			al+=4;	l+=2;
		}
		if (l==llim) {
			y0  = al[1]*cost*y1 + al[0]*y0;
			re += y0 * ql[l];
		}

		q[it] = re+ro;
		q[nlat_2*2-1-it] = re-ro;
	}
	*/
}

template<int S, int NFIELDS, typename real=double>
static void leg_m0(shtns_cfg shtns, const real *ql, real *q, const int llim, int spat_dist = 0)
{
	const int nlat_2 = shtns->nlat_2;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256;		// good value
	const int NW = 1;

	// Launch the Legendre CUDA Kernel
	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (spat_dist == 0) spat_dist = shtns->spat_stride;
	for (int f=0; f<NFIELDS; f++) {
		leg_m0_kernel<BLOCKSIZE, S,1,real> <<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, ql + f*shtns->nlm_stride, q + f*spat_dist, llim, nlat_2);
	}
}


/*
__inline__ __device__
void warp_reduce_add_4(double& re, double& ro, double& ie, double& io) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
	re += shfl_down(re, offset);
	ro += shfl_down(ro, offset);
	ie += shfl_down(ie, offset);
	io += shfl_down(io, offset);
  }
}

__inline__ __device__
void warp_reduce_add_2(double& ev, double& od) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
	ev += shfl_down(ev, offset);
	od += shfl_down(od, offset);
  }
}

__inline__ __device__
void warp_reduce_add(double& ev) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
	ev += shfl_down(ev, offset);
  }
}
*/

template<int BLOCKSIZE, int LSPAN, int S, int NFIELDS, typename real=double> __global__ void
ileg_m0_kernel(const real* __restrict__ al, const real* __restrict__ ct, const real* __restrict__ q, real *ql, const int llim, const int nlat_2, const int lmax, const int q_dist=0, const int ql_dist=0)
{
	const int it = BLOCKSIZE * blockIdx.x + threadIdx.x;
	const int j = threadIdx.x;

	// re-assign each thread an l (transpose)
	const int ll = j / (BLOCKSIZE/LSPAN);

	__shared__ real ak[2*LSPAN+2];	// cache
	__shared__ real yl[LSPAN*BLOCKSIZE];		// yl is also used for even/odd computation. Ensure LSPAN >= 4.
	const int l_inc = BLOCKSIZE;
	const real cost = (it < nlat_2) ? ct[it] : 0.0;
	real y0, y1;

	if (LSPAN < 4) printf("ERROR: LSPAN<4\n");

	real my_reo[NFIELDS][LSPAN];			// in registers
	if (j < 2*LSPAN+2) ak[j] = al[j];

	#pragma unroll
	for (int f=0; f<NFIELDS; f++) {
		y0 = (it < nlat_2) ? q[it + f*q_dist] : 0.0;				// north
		y1 = (it < nlat_2) ? q[nlat_2*2-1 - it + f*q_dist] : 0.0;	// south

		if ((f>0) && (BLOCKSIZE > WARPSZE)) 	__syncthreads();
		yl[j] = y0+y1;					// even
		yl[BLOCKSIZE +j] = y0-y1;		// odd
		if (BLOCKSIZE > WARPSZE) 	__syncthreads();

		// transpose reo to my_reo
		#pragma unroll
		for (int i=0, k=0; i<BLOCKSIZE; i+= BLOCKSIZE/LSPAN, k++) {
			int it = j % (BLOCKSIZE/LSPAN) + i;
			my_reo[f][k] = yl[(ll&1)*BLOCKSIZE +it];
		}
	}

	int l = 0;
	y0 = (it < nlat_2) ? ct[it + nlat_2] : 0.0;		// weights are stored just after ct.
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);
	y0 *= ak[0];
	y1 = y0 * ak[1] * cost;

	if (BLOCKSIZE > WARPSZE)	__syncthreads();
	
	yl[j] = y0;
	yl[l_inc +j] = y1;
	al+=2;
	while (l <= llim) {
		for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
			yl[k*l_inc +j]     = y0;
			y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
			yl[(k+1)*l_inc +j] = y1;
			y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
			al += 4;
		}
		if(BLOCKSIZE > WARPSZE)	__syncthreads();

		real qll[NFIELDS];	// accumulator
		// now re-assign each thread an l (transpose)
		const int itl = ll*l_inc + j % (BLOCKSIZE/LSPAN);
		#pragma unroll
		for (int f=0; f<NFIELDS; f++) qll[f] = my_reo[f][0] * yl[itl];			// first element
		#pragma unroll
		for (int i=BLOCKSIZE/LSPAN, k=1; i<BLOCKSIZE; i+= BLOCKSIZE/LSPAN, k++) {		// accumulate
			#pragma unroll
			for (int f=0; f<NFIELDS; f++)	qll[f] += my_reo[f][k] * yl[itl+i];
		}

		if (BLOCKSIZE/LSPAN <= WARPSZE) {	// reduce_add within same l is in same warp too:
			if (WARPSZE % (BLOCKSIZE/LSPAN)) printf("ERROR\n");
			#pragma unroll
			for (int ofs = BLOCKSIZE/(LSPAN*2); ofs > 0; ofs>>=1) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qll[f] += shfl_down(qll[f], ofs, BLOCKSIZE/LSPAN);
			}
			if ( ((j % (BLOCKSIZE/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
				if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	ql[2*(l+ll) + f*ql_dist] = qll[f];
				} else {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
				}
			}
		} else {	// only partial reduction possible, finish with atomicAdd():
			if ((BLOCKSIZE/LSPAN) % WARPSZE) printf("ERROR\n");
			#pragma unroll
			for (int ofs = WARPSZE/2; ofs > 0; ofs>>=1) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qll[f] += shfl_down(qll[f], ofs, WARPSZE);
			}
			__syncthreads();
			const int nsum = (BLOCKSIZE/(LSPAN*WARPSZE));
			if ((j % WARPSZE) == 0) {
				for (int f=0; f<NFIELDS; f++)  yl[ll*nsum + ((j/WARPSZE) % nsum) + f*LSPAN*nsum] = qll[f];
			}
			__syncthreads();
			if ( ((j % (BLOCKSIZE/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
				for (int i=1; i<nsum; i++) {
					for (int f=0; f<NFIELDS; f++)	qll[f] += yl[ll*nsum + i + f*LSPAN*nsum];
				}
				if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	ql[2*(l+ll) + f*ql_dist] = qll[f];
				} else {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
				}
			}
		/*	if ( ((j % WARPSZE) == 0) && ((l+ll)<=llim) ) {	// write result
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
			}*/
		}

		if (j<2*LSPAN) ak[j+2] = al[j];
		if (BLOCKSIZE > WARPSZE)	__syncthreads();
		l+=LSPAN;
	}
}

template<int S, int NFIELDS, typename real=double>
static void ileg_m0(shtns_cfg shtns, const real* q, real *ql, const int llim, int q_dist=0, int ql_dist=0)
{
	const int nlat_2 = shtns->nlat_2;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256/NFIELDS;
	const int LSPAN_ = 8/NFIELDS;
	const int NW = 1;

	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (q_dist == 0) q_dist = shtns->spat_stride;
	if (ql_dist == 0) ql_dist = shtns->nlm_stride;
	ileg_m0_kernel<BLOCKSIZE, LSPAN_, S, NFIELDS, real><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, q, ql, llim, nlat_2, q_dist, ql_dist);
}


/** \internal convert from vector SH to scalar SH
	Vlm =  st*d(Slm)/dtheta + I*m*Tlm
	Wlm = -st*d(Tlm)/dtheta + I*m*Slm
*/
template<int BLOCKSIZE, typename real=double> __global__ void
sphtor2scal_kernel(const double* __restrict__ mx, const real* __restrict__ slm, const real* __restrict__ tlm, real *vlm, real *wlm, const int llim, const int lmax, const int mres)
{
	// indices for overlapping blocks:
	const int ll = (blockDim.x-4) * blockIdx.x + threadIdx.x - 2;		// = 2*l + ((imag) ? 1 : 0)
	const int j = threadIdx.x;
	const int im = blockIdx.y;

	__shared__ real sl[BLOCKSIZE];
	__shared__ real tl[BLOCKSIZE];
	__shared__ real M[BLOCKSIZE];

	const int m = im*mres;
	//int ofs = im*(2*(lmax+1) -m + mres);
//    const int xchg = 1 - 2*(ll&1);	// +1 for real and -1 for imag
//	const int xchg = ll - (ll^1);	// -1 for real and +1 for imag
	const int ofs = im*(((lmax+1)<<1) -m + mres) + ll;

	if ( (ll >= 0) && (ll < 2*(llim+1-m)) ) {
		M[j] = mx[ofs];
		sl[j] = slm[ofs];
		tl[j] = tlm[ofs];
	} else {
		M[j] = 0.0;
		sl[j] = 0.0;
		tl[j] = 0.0;
	}
	const real mimag = im * mres * (ll - (ll^1));

	__syncthreads();

//    if ((j>=2) && (j<BLOCKSIZE-2) && (ll < 2*(llim+2-m))) {
	if ((j<BLOCKSIZE-4) && (ll < 2*(llim+1-m))) {
		real ml = M[2*(j>>1)+1];
		real mu = M[2*(j>>1)+2];
		real v = mimag*tl[(j+2)^1]  +  (ml*sl[j] + mu*sl[j+4]);
		real w = mimag*sl[(j+2)^1]  -  (ml*tl[j] + mu*tl[j+4]);
		vlm[ofs+2*im+2] = v;
		wlm[ofs+2*im+2] = w;
	}
}

/** \internal convert from 2 scalar SH to vector SH
	Slm = - (I*m*Wlm + MX*Vlm) / (l*(l+1))
	Tlm = - (I*m*Vlm - MX*Wlm) / (l*(l+1))
**/
template<int BLOCKSIZE, typename real=double> __global__ void
scal2sphtor_kernel(const double* __restrict__ mx, const real* __restrict__ vlm, const real* __restrict__ wlm, real *slm, real *tlm, const int llim, const int lmax, const int mres)
{
	// indices for overlapping blocks:
	const int ll = (blockDim.x-4) * blockIdx.x + threadIdx.x - 2;		// = 2*l + ((imag) ? 1 : 0)
	const int j = threadIdx.x;
	const int im = blockIdx.y;

	__shared__ real vl[BLOCKSIZE];
	__shared__ real wl[BLOCKSIZE];
	__shared__ real M[BLOCKSIZE];

	const int m = im * mres;
	//const int xchg = 1 - 2*(j&1);	// +1 for real and -1 for imag
	//const int xchg = (j^1) - j;		// +1 for real and -1 for imag
	int ofs = im*(2*(lmax+1) -m + mres)  + ll;

	if ( (ll >= 0) && (ll < 2*(llim+1-m)) ) {
		M[j] = mx[ofs];
	} else M[j] = 0.0;

	if ( (ll >= 0) && (ll < 2*(llim+2-m)) ) {
		vl[j] = vlm[ofs+2*im];
		wl[j] = wlm[ofs+2*im];
	} else {
		vl[j] = 0.0;
		wl[j] = 0.0;
	}

	int ell = (ll>>1) + m + 1;		// +1 because we shift below

	__syncthreads();

//    if ((j>=2) && (j<THREADS_PER_BLOCK-2) && (ll < 2*(llim+1-m))) {
	if (j<BLOCKSIZE-4) {
		if ((ell <= llim) && (ell>0)) {
			const real mimag = im * mres * ((j^1) -j);
			real ll_1 = 1.0 / (ell*(ell+1));
			real ml = M[2*(j>>1)+1];
			real mu = M[2*(j>>1)+2];
			real s = mimag*wl[(j+2)^1]  -  (ml*vl[j] + mu*vl[j+4]);
			real t = mimag*vl[(j+2)^1]  +  (ml*wl[j] + mu*wl[j+4]);
			slm[ofs+2] = s * ll_1;
			tlm[ofs+2] = t * ll_1;
		} else if (ell <= lmax) {	// fill with zeros up to lmax (and l=0 too).
			slm[ofs+2] = 0.0;
			tlm[ofs+2] = 0.0;
		}
	}
}

void sphtor2scal_gpu(shtns_cfg shtns, cplx* d_Slm, cplx* d_Tlm, cplx* d_Vlm, cplx* d_Wlm, int llim, int mmax)
{
	dim3 blocks((2*(shtns->lmax+2)+MAX_THREADS_PER_BLOCK-5)/(MAX_THREADS_PER_BLOCK-4), mmax+1);
	dim3 threads(MAX_THREADS_PER_BLOCK, 1);
	sphtor2scal_kernel<MAX_THREADS_PER_BLOCK> <<< blocks, threads,0, shtns->comp_stream >>>
		(shtns->d_mx_stdt, (double*) d_Slm, (double*) d_Tlm, (double*) d_Vlm, (double*) d_Wlm, llim, shtns->lmax, shtns->mres);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) { printf("sphtor2scal_gpu error : %s!\n", hipGetErrorString(err));	return; }
}

void scal2sphtor_gpu(shtns_cfg shtns, cplx* d_Vlm, cplx* d_Wlm, cplx* d_Slm, cplx* d_Tlm, int llim)
{
	dim3 blocks((2*(shtns->lmax+2)+MAX_THREADS_PER_BLOCK-5)/(MAX_THREADS_PER_BLOCK-4), shtns->mmax+1);
	dim3 threads(MAX_THREADS_PER_BLOCK, 1);
	scal2sphtor_kernel<MAX_THREADS_PER_BLOCK> <<<blocks, threads, 0, shtns->comp_stream>>>
		(shtns->d_mx_van, (double*) d_Vlm, (double*) d_Wlm, (double*)d_Slm, (double*)d_Tlm, llim, shtns->lmax, shtns->mres);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) { printf("scal2sphtor_gpu error : %s!\n", hipGetErrorString(err));	return; }
}



/// requirements : blockSize must be 1 in the y-direction and THREADS_PER_BLOCK in the x-direction.
/// llim MUST BE <= 1800
/// S can only be 0 (for scalar) or 1 (for spin 1 / vector)
template<int BLOCKSIZE, int S, int NFIELDS, int NW, typename real=double>
static __global__ void leg_m_lowllim_kernel(
	const real* __restrict__ al, const real* __restrict__ ct, const real* __restrict__ ql, real *q,
	const int llim, const int nlat_2, const int lmax, const int mres, const int nphi, const int ql_dist=0, const int q_dist=0)
{
	const int it = BLOCKSIZE*NW * blockIdx.x + threadIdx.x;
	const int im = blockIdx.y;
	const int j = threadIdx.x;
	const int m_inc = 2*nlat_2;
	const int k_inc = 1;

	__shared__ real ak[BLOCKSIZE];		// size blockDim.x
	__shared__ real qk[NFIELDS][BLOCKSIZE];	// size blockDim.x * NFIELDS

	real cost[NW];
	real y0[NW];
	real y1[NW];
	#pragma unroll
	for (int i=0; i<NW; i++) {
		const int iit = it+i*BLOCKSIZE;
		cost[i] = (iit < nlat_2) ? ct[iit] : 0.0;
	}

	if (im==0) {
		ak[j] = al[j+2];
		if (j<2*(llim+1)) {
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) 	qk[f][j] = ql[j  + f*ql_dist];
		}
		real re[NFIELDS][NW], ro[NFIELDS][NW];
		#pragma unroll
		for (int f=0; f<NFIELDS; f++) {
			#pragma unroll
			for (int i=0; i<NW; i++) {
				re[f][i] = 0.0;
				ro[f][i] = 0.0;
			}
		}
		int l = 0;
		for (int i=0; i<NW; i++) y0[i] = al[0];
		if (S==1) for (int i=0; i<NW; i++) y0[i] *= rsqrt(1.0 - cost[i]*cost[i]);	// for vectors, divide by sin(theta)
		for (int i=0; i<NW; i++) y1[i] = y0[i] * al[1] * cost[i];
		al+=2;
		__syncthreads();
		while(l<=llim-BLOCKSIZE/2) {
			#pragma unroll
			for (int k=0; k<BLOCKSIZE; k+=4) {
				#pragma unroll
				for (int i=0; i<NW; i++) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	re[f][i] += y0[i] * qk[f][k];
				}
				#pragma unroll
				for (int i=0; i<NW; i++) 	y0[i] = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
				#pragma unroll
				for (int i=0; i<NW; i++) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	ro[f][i] += y1[i] * qk[f][k+2];
				}
				#pragma unroll
				for (int i=0; i<NW; i++)	y1[i] = ak[k+3]*cost[i]*y0[i] + ak[k+2]*y1[i];
			}
			al += BLOCKSIZE;
			l += BLOCKSIZE/2;
			__syncthreads();
			if (l+j/2 <= llim) {
				ak[j] = al[j];
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qk[f][j] = ql[2*l+j + f*ql_dist];
			}
			__syncthreads();
		}
		int k=0;
		while(l<llim) {
			#pragma unroll
			for (int i=0; i<NW; i++) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	re[f][i] += y0[i] * qk[f][k];
			}
			#pragma unroll
			for (int i=0; i<NW; i++) 	y0[i]  = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
			#pragma unroll
			for (int i=0; i<NW; i++) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	ro[f][i] += y1[i] * qk[f][k+2];
			}
			#pragma unroll
			for (int i=0; i<NW; i++)	y1[i]  = ak[k+3]*cost[i]*y0[i] + ak[k+2]*y1[i];
			l+=2;	  k+=4;
		}
		if (l==llim) {
			#pragma unroll
			for (int i=0; i<NW; i++) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	re[f][i] += y0[i] * qk[f][k];
			}
		}
		#pragma unroll
		for (int i=0; i<NW; i++) {
			const int iit = it+i*BLOCKSIZE;
			if (iit < nlat_2) {
				// store mangled for complex fft
				const int iit = it+i*BLOCKSIZE;
				#pragma unroll
				for (int f=0; f<NFIELDS; f++) {
					q[iit*k_inc + f*q_dist] = re[f][i]+ro[f][i];
					q[(nlat_2*2-1-iit)*k_inc + f*q_dist] = re[f][i]-ro[f][i];
				}
			}
		}
	} else { 	// m>0
		real rer[NFIELDS][NW], ror[NFIELDS][NW], rei[NFIELDS][NW], roi[NFIELDS][NW];
		int m = im*mres;
		int l = (im*(2*(lmax+1)-(m+mres)))>>1;
		al += 2*(l+m);
		ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

		#pragma unroll
		for (int i=0; i<NW; i++) 	y1[i] = sqrt(1.0 - cost[i]*cost[i]);		// y1 = sin(theta)
		ak[j] = al[j+2];
		#pragma unroll
		for (int f=0; f<NFIELDS; f++)	if (m+j/2 <= llim) qk[f][j] = ql[2*m+j + f*ql_dist];

		#pragma unroll
		for (int i=0; i<NW; i++) {
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) {
				ror[f][i] = 0.0;		roi[f][i] = 0.0;
				rer[f][i] = 0.0;		rei[f][i] = 0.0;
			}
			y0[i] = 1.0;
		}
		l = m - S;
		do {		// sin(theta)^(m-S)
			if (l&1) {
				#pragma unroll
				for (int i=0; i<NW; i++) y0[i] *= y1[i];
			}
			#pragma unroll
			for (int i=0; i<NW; i++) y1[i] *= y1[i];
		} while(l >>= 1);

		#pragma unroll
		for (int i=0; i<NW; i++) y0[i] *= al[0];
		#pragma unroll
		for (int i=0; i<NW; i++) y1[i] = al[1]*y0[i]*cost[i];

		__syncthreads();
		l=m;		al+=2;
		while (l<=llim - BLOCKSIZE/2) {	// compute even and odd parts
			#pragma unroll
			for (int k = 0; k<BLOCKSIZE; k+=4) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++) {
					#pragma unroll
					for (int i=0; i<NW; i++) {
						rer[f][i] += y0[i] * qk[f][k];		// real
						rei[f][i] += y0[i] * qk[f][k+1];	// imag
					}
				}
				#pragma unroll
				for (int i=0; i<NW; i++) 	y0[i] = ak[k+1]*(cost[i]*y1[i]) + ak[k]*y0[i];
				#pragma unroll
				for (int f=0; f<NFIELDS; f++) {
					#pragma unroll
					for (int i=0; i<NW; i++) {
						ror[f][i] += y1[i] * qk[f][k+2];	// real
						roi[f][i] += y1[i] * qk[f][k+3];	// imag
					}
				}
				#pragma unroll
				for (int i=0; i<NW; i++) 	y1[i] = ak[k+3]*(cost[i]*y0[i]) + ak[k+2]*y1[i];
			}
			al += BLOCKSIZE;
			l += BLOCKSIZE/2;
			__syncthreads();
			if (l+j/2 <= llim) {
				ak[j] = al[j];
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qk[f][j] = ql[2*l+j + f*ql_dist];
			}
			__syncthreads();
		}
		int k=0;
		while (l<llim) {	// compute even and odd parts
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) {
				#pragma unroll
				for (int i=0; i<NW; i++) {
					rer[f][i] += y0[i] * qk[f][k];		// real
					rei[f][i] += y0[i] * qk[f][k+1];	// imag
				}
			}
			#pragma unroll
			for (int i=0; i<NW; i++) 	y0[i] = ak[k+1]*(cost[i]*y1[i]) + ak[k]*y0[i];
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) {
				#pragma unroll
				for (int i=0; i<NW; i++) {
					ror[f][i] += y1[i] * qk[f][k+2];	// real
					roi[f][i] += y1[i] * qk[f][k+3];	// imag
				}
			}
			#pragma unroll
			for (int i=0; i<NW; i++) 	y1[i] = ak[k+3]*(cost[i]*y0[i]) + ak[k+2]*y1[i];
			l+=2;	k+=4;
		}
		if (l==llim) {
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) {
				#pragma unroll
				for (int i=0; i<NW; i++) {
					rer[f][i] += y0[i] * qk[f][k];		// real
					rei[f][i] += y0[i] * qk[f][k+1];	// imag
				}
			}
		}

		/// store mangled for complex fft
		#pragma unroll
		for (int i=0; i<NW; i++) {
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) {
				rei[f][i] = shfl_xor(rei[f][i], 1);
				roi[f][i] = shfl_xor(roi[f][i], 1);
			}
		}
		real nr[NFIELDS][NW];
		const real sgn = 1 - 2*(j&1);
		#pragma unroll
		for (int i=0; i<NW; i++) {
			const int iit = it+i*BLOCKSIZE;
			if (iit < nlat_2) {
				#pragma unroll
				for (int f=0; f<NFIELDS; f++) {
					nr[f][i] =  rer[f][i]+ror[f][i];
					rer[f][i] = rer[f][i]-ror[f][i];
					ror[f][i] = sgn*(rei[f][i]+roi[f][i]);
					rei[f][i] = sgn*(rei[f][i]-roi[f][i]);
				}
				#pragma unroll
				for (int f=0; f<NFIELDS; f++) {
					q[im*m_inc + iit*k_inc + f*q_dist]                     = nr[f][i]  - ror[f][i];
					q[(nphi-im)*m_inc + iit*k_inc + f*q_dist]              = nr[f][i]  + ror[f][i];
					q[im*m_inc + (nlat_2*2-1-iit)*k_inc + f*q_dist]        = rer[f][i] + rei[f][i];
					q[(nphi-im)*m_inc + (nlat_2*2-1-iit)*k_inc + f*q_dist] = rer[f][i] - rei[f][i];
				}
			}
		}
	}
}

template<int S, int NFIELDS, typename real=double>
static void leg_m_lowllim(shtns_cfg shtns, const real *ql, real *q, const int llim, const int mmax, int spat_dist=0)
{
	const int lmax = shtns->lmax;
	const int mres = shtns->mres;
	const int nlat_2 = shtns->nlat_2;
	const int nphi = shtns->nphi;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256;		// good value
	const int NW = 2;

	// Launch the Legendre CUDA Kernel
	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (spat_dist == 0) spat_dist = shtns->spat_stride;
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	leg_m_lowllim_kernel<BLOCKSIZE, S, NFIELDS, NW, real> <<<blocks, threads, 0, stream>>>(d_alm, d_ct, (real*) ql, (real*) q, llim, nlat_2, lmax,mres, nphi, shtns->nlm_stride, spat_dist);
}

/// requirements : blockSize must be 1 in the y-direction and THREADS_PER_BLOCK in the x-direction.
/// llim can be arbitrarily large (> 1800)
template<int BLOCKSIZE, int S, typename real=double> __global__ void
leg_m_highllim_kernel(const real *al, const real *ct, const real *ql, real *q, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi)
{
	const int it = blockDim.x * blockIdx.x + threadIdx.x;
	const int im = blockIdx.y;
	const int j = threadIdx.x;
	const int m_inc = 2*nlat_2;
	const int k_inc = 1;
	const real accuracy = (sizeof(real) >= 8) ? SHT_ACCURACY : SHT_ACCURACY_FLOAT;
	const real scale_factor = (sizeof(real) >= 8) ? SHT_SCALE_FACTOR : SHT_SCALE_FACTOR_FLOAT;

	__shared__ real ak[BLOCKSIZE];	// cache
	__shared__ real qk[BLOCKSIZE];

	const real cost = (it < nlat_2) ? ct[it] : 0.0;

	if (im==0) {
	int l = 0;
	real y0 = al[0];
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);
	real re = y0 * ql[0];
	real y1 = y0 * al[1] * cost;
	real ro = y1 * ql[2];
	al+=2;    l+=2;
	while(l<llim) {
		y0  = al[1]*(cost*y1) + al[0]*y0;
		re += y0 * ql[2*l];
		y1  = al[3]*(cost*y0) + al[2]*y1;
		ro += y1 * ql[2*l+2];
		al+=4;	l+=2;
	}
	if (l==llim) {
		y0  = al[1]*cost*y1 + al[0]*y0;
		re += y0 * ql[2*l];
	}
	if (it < nlat_2) {
		// store mangled for complex fft
		q[it*k_inc] = re+ro;
		q[(nlat_2*2-1-it)*k_inc] = re-ro;
	}
	} else { 	// m>0
	int m = im*mres;
	int l = (im*(2*(lmax+1)-(m+mres)))>>1;
	al += 2*(l+m);
	ql += 2*(l + S*im);
	real rer,ror, rei,roi, y0, y1;
	ror = 0.0;	roi = 0.0;
	rer = 0.0;	rei = 0.0;
	y1 = sqrt(1.0 - cost*cost);	// sin(theta)
	if (_any(m - llim*y1 <= max(80, llim>>7))) {		// polar optimization (see Reinecke 2013), avoiding warp divergence
		y0 = 1.0;	// y0
		l = m - S;
		int ny = 0;
		int nsint = 0;
		do {		// sin(theta)^(m-S)		(use rescaling to avoid underflow)
		if (l&1) {
			y0 *= y1;
			ny += nsint;
			if (y0 < (accuracy+1.0/scale_factor)) {		// possible warp divergence
			ny--;
			y0 *= scale_factor;
			}
		}
		y1 *= y1;
		nsint += nsint;
		if (y1 < 1.0/scale_factor) {		// possible warp divergence
			nsint--;
			y1 *= scale_factor;
		}
		} while(l >>= 1);
		y0 *= al[0];
		y1 = 0.0;
//	    y1 = al[1]*y0*cost;

		l=m;	int ka = WARPSZE;
		const int ofs = j & 0xFFE0;

		while ( _all(ny<0) && (l<llim) ) {
			if (ka+4 >= WARPSZE) {
				ak[j] = al[(j&31)];
				ka=0;
			}
			y1 = ak[ka+1+ofs]*cost*y0 + ak[ka+ofs]*y1;
			y0 = ak[ka+3+ofs]*cost*y1 + ak[ka+2+ofs]*y0;
			l+=2;	al+=4;	ka+=4;
			if (fabs(y1) > accuracy*scale_factor + 1.0)
			{	// rescale when value is significant
				++ny;
				y0 *= 1.0/scale_factor;
				y1 *= 1.0/scale_factor;
			}
		}

		ka = WARPSZE;
		while (l<llim) {
			if (ka+4 >= WARPSZE) {		// cache coefficients
				ak[j] = al[(j&31)];
				qk[j] = ql[2*l+(j&31)];
				ka = 0;
			}
			y1 = ak[ka+1+ofs]*cost*y0 + ak[ka+ofs]*y1;
			if (ny==0) {
				rer += y0 * qk[ka+ofs];	// real
				rei += y0 * qk[ka+1+ofs];	// imag
				ror += y1 * qk[ka+2+ofs];	// real
				roi += y1 * qk[ka+3+ofs];	// imag
			}
			else if (fabs(y0) > accuracy*scale_factor + 1.0)
			{	// rescale when value is significant
				++ny;
				y0 *= 1.0/scale_factor;
				y1 *= 1.0/scale_factor;
			}
			l+=2;	al+=4;
			y0 = ak[ka+3+ofs]*cost*y1 + ak[ka+2+ofs]*y0;
			ka+=4;
		}
		if ((l==llim) && (ny==0)) {
			rer += y0 * ql[2*l];
			rei += y0 * ql[2*l+1];
		}
	}

	/// store mangled for complex fft
	real nr = rer+ror;
	real sr = rer-ror;
	const real sgn = 1 - 2*(j&1);
	rei = shfl_xor(rei, 1);
	roi = shfl_xor(roi, 1);
	real nix = sgn*(rei+roi);
	real six = sgn*(rei-roi);
	if (it < nlat_2) {
		q[im*m_inc + it*k_inc]                     = nr - nix;
		q[(nphi-im)*m_inc + it*k_inc]              = nr + nix;
		q[im*m_inc + (nlat_2*2-1-it)*k_inc]        = sr + six;
		q[(nphi-im)*m_inc + (nlat_2*2-1-it)*k_inc] = sr - six;
	}
	}
}

template<int S, int NFIELDS, typename real=double>
static void leg_m_highllim(shtns_cfg shtns, const real *ql, real *q, const int llim, const int mmax, int spat_dist = 0)
{
	const int lmax = shtns->lmax;
	const int mres = shtns->mres;
	const int nlat_2 = shtns->nlat_2;
	const int nphi = shtns->nphi;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256;		// good value
	const int NW = 1;

	// Launch the Legendre CUDA Kernel
	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (spat_dist == 0) spat_dist = shtns->spat_stride;
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	for (int f=0; f<NFIELDS; f++) {
		leg_m_highllim_kernel<BLOCKSIZE, S,real> <<<blocks, threads, 0, stream>>>(d_alm, d_ct, ql + f*shtns->nlm_stride, q + f*spat_dist, llim, nlat_2, lmax,mres, nphi);
	}
}


template<int BLOCKSIZE, int LSPAN, int S, int NFIELDS, typename real=double> __global__ void
ileg_m_lowllim_kernel(const real* __restrict__ al, const real* __restrict__ ct, const real* __restrict__ q, real *ql, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi, const real mpos_scale, const int q_dist=0, const int ql_dist=0)
{
	const int it = BLOCKSIZE * blockIdx.x + threadIdx.x;
	const int j = threadIdx.x;
	const int im = blockIdx.y;
	const int m_inc = 2*nlat_2;
//    const int k_inc = 1;

	// re-assign each thread an l (transpose)
	const int ll = j / (BLOCKSIZE/LSPAN);
	const int ri = j / (BLOCKSIZE/(2*LSPAN)) % 2;	// real (0) or imag (1)

	__shared__ real ak[2*LSPAN+2];	// cache
	__shared__ real yl[LSPAN*BLOCKSIZE];		// yl is also used for even/odd computation. Ensure LSPAN >= 4.
	const int l_inc = BLOCKSIZE;
	const real cost = (it < nlat_2) ? ct[it] : 0.0;
	real y0, y1;

	if (LSPAN < 4) printf("ERROR: LSPAN<4\n");

	if (im == 0) {
		real my_reo[NFIELDS][LSPAN];			// in registers
		if (j < 2*LSPAN+2) ak[j] = al[j];

		#pragma unroll
		for (int f=0; f<NFIELDS; f++) {
			y0 = (it < nlat_2) ? q[it + f*q_dist] : 0.0;				// north
			y1 = (it < nlat_2) ? q[nlat_2*2-1 - it + f*q_dist] : 0.0;	// south

			if ((f>0) && (BLOCKSIZE > WARPSZE)) 	__syncthreads();
			yl[j] = y0+y1;					// even
			yl[BLOCKSIZE +j] = y0-y1;		// odd
			if (BLOCKSIZE > WARPSZE) 	__syncthreads();

			// transpose reo to my_reo
			#pragma unroll
			for (int i=0, k=0; i<BLOCKSIZE; i+= BLOCKSIZE/LSPAN, k++) {
				int it = j % (BLOCKSIZE/LSPAN) + i;
				my_reo[f][k] = yl[(ll&1)*BLOCKSIZE +it];
			}
		}

		int l = 0;
		y0 = (it < nlat_2) ? ct[it + nlat_2] : 0.0;		// weights are stored just after ct.
		if (S==1) y0 *= rsqrt(1.0 - cost*cost);
		y0 *= ak[0];
		y1 = y0 * ak[1] * cost;

		if (BLOCKSIZE > WARPSZE)	__syncthreads();
		
		yl[j] = y0;
		yl[l_inc +j] = y1;
		al+=2;
		while (l <= llim) {
			for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
				yl[k*l_inc +j]     = y0;
				y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
				yl[(k+1)*l_inc +j] = y1;
				y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
				al += 4;
			}
			if(BLOCKSIZE > WARPSZE)	__syncthreads();

			real qll[NFIELDS];	// accumulator
			// now re-assign each thread an l (transpose)
			const int itl = ll*l_inc + j % (BLOCKSIZE/LSPAN);
			#pragma unroll
			for (int f=0; f<NFIELDS; f++) qll[f] = my_reo[f][0] * yl[itl];			// first element
			#pragma unroll
			for (int i=BLOCKSIZE/LSPAN, k=1; i<BLOCKSIZE; i+= BLOCKSIZE/LSPAN, k++) {		// accumulate
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qll[f] += my_reo[f][k] * yl[itl+i];
			}

			if (BLOCKSIZE/LSPAN <= WARPSZE) {	// reduce_add within same l is in same warp too:
				if (WARPSZE % (BLOCKSIZE/LSPAN)) printf("ERROR\n");
				#pragma unroll
				for (int ofs = BLOCKSIZE/(LSPAN*2); ofs > 0; ofs>>=1) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	qll[f] += shfl_down(qll[f], ofs, BLOCKSIZE/LSPAN);
				}
				if ( ((j % (BLOCKSIZE/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
					if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	ql[2*(l+ll) + f*ql_dist] = qll[f];
					} else {
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
					}
				}
			} else {	// only partial reduction possible, finish with atomicAdd():
				if ((BLOCKSIZE/LSPAN) % WARPSZE) printf("ERROR\n");
				#pragma unroll
				for (int ofs = WARPSZE/2; ofs > 0; ofs>>=1) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	qll[f] += shfl_down(qll[f], ofs, WARPSZE);
				}
				__syncthreads();
				const int nsum = (BLOCKSIZE/(LSPAN*WARPSZE));
				if ((j % WARPSZE) == 0) {
					for (int f=0; f<NFIELDS; f++)  yl[ll*nsum + ((j/WARPSZE) % nsum) + f*LSPAN*nsum] = qll[f];
				}
				__syncthreads();
				if ( ((j % (BLOCKSIZE/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
					for (int i=1; i<nsum; i++) {
						for (int f=0; f<NFIELDS; f++)	qll[f] += yl[ll*nsum + i + f*LSPAN*nsum];
					}
					if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	ql[2*(l+ll) + f*ql_dist] = qll[f];
					} else {
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
					}
				}
			/*	if ( ((j % WARPSZE) == 0) && ((l+ll)<=llim) ) {	// write result
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll) + f*ql_dist, qll[f]);		// VERY slow atomic add on Kepler.
				}*/
			}

			if (j<2*LSPAN) ak[j+2] = al[j];
			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			l+=LSPAN;
		}
	} else {	// im > 0
		real my_reo[NFIELDS][2*LSPAN];			// in registers
		int m = im*mres;
		int l = (im*(2*(lmax+1)-(m+mres)))>>1;
		al += 2*(l+m);
		ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

		if (j < 2*LSPAN+2) ak[j] = al[j];
		const real sgn = 2*(j&1) - 1;	// -/+
		
		#pragma unroll
		for (int f=0; f<NFIELDS; f++) {
			y0         = (it < nlat_2) ? q[im*m_inc + it + f*q_dist] : 0.0;		// north imag (ani)
			real qer = (it < nlat_2) ? q[(nphi-im)*m_inc + it + f*q_dist] : 0.0;	// north real (an)
			y1         = (it < nlat_2) ? q[im*m_inc + nlat_2*2-1-it + f*q_dist] : 0.0;	// south imag (asi)
			real qor = (it < nlat_2) ? q[(nphi-im)*m_inc + nlat_2*2-1-it + f*q_dist] : 0.0;	// south real (as)
			real qei = y0-qer;		qer += y0;		// ani = -qei[lane+1],   bni = qei[lane-1]
			real qoi = y1-qor;		qor += y1;		// bsi = -qoi[lane-1],   asi = qoi[lane+1];
			y0 = shfl_xor(qei, 1);	// exchange between adjacent lanes.
			y1 = shfl_xor(qoi, 1);

			if ((f>0) && (BLOCKSIZE > WARPSZE)) 	__syncthreads();

			yl[j] 		       = qer + qor;	// rer
			yl[BLOCKSIZE +j]   = qer - qor;	// ror
			yl[2*BLOCKSIZE +j] = sgn*(y0 - y1);	// rei
			yl[3*BLOCKSIZE +j] = sgn*(y0 + y1);	// roi

			if (BLOCKSIZE > WARPSZE) 	__syncthreads();
			// transpose yl to my_reo
			#pragma unroll
			for (int i=0, k=0; i<BLOCKSIZE; i+= BLOCKSIZE/(2*LSPAN), k++) {
				int it = j % (BLOCKSIZE/(2*LSPAN)) + i;
				my_reo[f][k] = yl[((ll&1)+2*ri)*BLOCKSIZE +it];
			}
		}

		y1 = sqrt(1.0 - cost*cost);	// sin(theta)

		y0 = mpos_scale * ak[0];	// y0
		l = m - S;
		do {		// sin(theta)^(m-S)
		if (l&1) y0 *= y1;
		y1 *= y1;
		} while(l >>= 1);
		if (it < nlat_2)     y0 *= ct[it + nlat_2];		// include quadrature weights.
		y1 = ak[1]*y0*cost;

		l=m;		al+=2;
		while (l <= llim) {
			if (BLOCKSIZE > WARPSZE) 	__syncthreads();
			for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
				yl[k*l_inc +j]     = y0;
				y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
				yl[(k+1)*l_inc +j] = y1;
				y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
				al += 4;
			}

			// transposed work:
			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			real qlri[NFIELDS];	// accumulator
			const int itl = ll*l_inc + j % (BLOCKSIZE/(2*LSPAN));
			#pragma unroll
			for (int f=0; f<NFIELDS; f++)	qlri[f] = my_reo[f][0] * yl[itl];		// first element
			#pragma unroll
			for (int i=BLOCKSIZE/(2*LSPAN), k=1; i<BLOCKSIZE; i+= BLOCKSIZE/(2*LSPAN),k++) {		// accumulate
				#pragma unroll
				for (int f=0; f<NFIELDS; f++)	qlri[f] += my_reo[f][k] * yl[itl + i];
			}

			
			if (BLOCKSIZE/(2*LSPAN) <= WARPSZE) {		// reduce_add within same l is in same warp too:
				if (WARPSZE % (BLOCKSIZE/(2*LSPAN))) printf("ERROR\n");
				#pragma unroll
				for (int ofs = BLOCKSIZE/(LSPAN*4); ofs > 0; ofs>>=1) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	qlri[f] += shfl_down(qlri[f], ofs, BLOCKSIZE/(LSPAN*2));
				}
				if ( ((j % (BLOCKSIZE/(2*LSPAN))) == 0) && ((l+ll)<=llim) ) {	// write result
					if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	ql[2*(l+ll)+ri + f*ql_dist]   = qlri[f];
					} else {
						#pragma unroll
						for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll)+ri + f*ql_dist, qlri[f]);		// VERY slow atomic add on Kepler.
					}
				}
			} else {	// only partial reduction possible, finish with atomicAdd():
				if ((BLOCKSIZE/(2*LSPAN)) % WARPSZE) printf("ERROR\n");
				#pragma unroll
				for (int ofs = WARPSZE; ofs > 0; ofs>>=1) {
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	qlri[f] += shfl_down(qlri[f], ofs, WARPSZE);
				}
				if ( ((j % WARPSZE) == 0) && ((l+ll)<=llim) ) {	// write result
					#pragma unroll
					for (int f=0; f<NFIELDS; f++)	atomicAdd(ql+2*(l+ll)+ri + f*ql_dist, qlri[f]);		// VERY slow atomic add on Kepler.
				}
			}

			if (j<2*LSPAN) ak[j+2] = al[j];
			l+=LSPAN;
		}
	}
}

template<int S, int NFIELDS, typename real=double>
static void ileg_m_lowllim(shtns_cfg shtns, const real* q, real *ql, const int llim, int q_dist=0, int ql_dist=0)
{
	const int lmax = shtns->lmax;
	const int mres = shtns->mres;
	const int nlat_2 = shtns->nlat_2;
	const int nphi = shtns->nphi;
	int mmax = shtns->mmax;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256/NFIELDS;
	const int LSPAN_ = 8/NFIELDS;
	const int NW = 1;

	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (q_dist == 0) q_dist = shtns->spat_stride;
	if (ql_dist == 0) ql_dist = shtns->nlm_stride;
	if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	ileg_m_lowllim_kernel<BLOCKSIZE, LSPAN_, S, NFIELDS, real><<<blocks, threads, 0, stream>>>(d_alm, d_ct, (real*) q, (real*) ql, llim, nlat_2, lmax,mres, nphi, shtns->mpos_scale_analys, q_dist, ql_dist);
}


template<int BLOCKSIZE, int LSPAN, int S, typename real=double> __global__ void
ileg_m_highllim_kernel(const real *al, const real *ct, const real *q, real *ql, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi, const real mpos_scale)
{
	const int it = BLOCKSIZE * blockIdx.x + threadIdx.x;
	const int j = threadIdx.x;
	const int im = blockIdx.y;
	const int m_inc = 2*nlat_2;
//    const int k_inc = 1;
	const real accuracy = (sizeof(real) >= 8) ? SHT_ACCURACY : SHT_ACCURACY_FLOAT;
	const real scale_factor = (sizeof(real) >= 8) ? SHT_SCALE_FACTOR : SHT_SCALE_FACTOR_FLOAT;

	__shared__ real ak[2*LSPAN+2];	// cache
	__shared__ real yl[LSPAN*BLOCKSIZE];
	__shared__ real reo[4*BLOCKSIZE];
	const int l_inc = BLOCKSIZE;
	const real cost = (it < nlat_2) ? ct[it] : 0.0;
	real y0, y1;


	if (im == 0) {
		if (j < 2*LSPAN+2) ak[j] = al[j];
		if (BLOCKSIZE > WARPSZE)	__syncthreads();
		y0 = (it < nlat_2) ? q[it] : 0.0;		// north
		y1 = (it < nlat_2) ? q[nlat_2*2-1 - it] : 0.0;	// south
		reo[j] = y0+y1;				// even
		reo[BLOCKSIZE +j] = y0-y1;		// odd

		int l = 0;
		y0 = (it < nlat_2) ? ct[it + nlat_2] : 0.0;		// weights are stored just after ct.
		if (S==1) y0 *= rsqrt(1.0 - cost*cost);
		y0 *= ak[0];
		y1 = y0 * ak[1] * cost;
		yl[j] = y0;
		yl[l_inc +j] = y1;
		al+=2;
		while (l <= llim) {
			for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
				yl[k*l_inc +j]     = y0;
				y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
				yl[(k+1)*l_inc +j] = y1;
				y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
				al += 4;
			}
			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			real qll = 0.0;	// accumulator
			// now re-assign each thread an l (transpose)
			const int ll = j / (BLOCKSIZE/LSPAN);
			for (int i=0; i<BLOCKSIZE; i+= BLOCKSIZE/LSPAN) {
				int it = j % (BLOCKSIZE/LSPAN) + i;
				qll += reo[(ll&1)*BLOCKSIZE +it] * yl[ll*l_inc +it];
			}

			// reduce_add within same l must be in same warp too:
			if (BLOCKSIZE/LSPAN > WARPSZE) printf("ERROR\n");

			for (int ofs = BLOCKSIZE/(LSPAN*2); ofs > 0; ofs>>=1) {
				qll += shfl_down(qll, ofs, BLOCKSIZE/LSPAN);
			}
			if ( ((j % (BLOCKSIZE/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
				if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
					ql[2*(l+ll)] = qll;
				} else {
					atomicAdd(ql+2*(l+ll), qll);		// VERY slow atomic add on Kepler.
				}
			}
			if (j<2*LSPAN) ak[j+2] = al[j];
			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			l+=LSPAN;
		}
	} else {	// im > 0
		int m = im*mres;
		int l = (im*(2*(lmax+1)-(m+mres)))>>1;
		al += 2*(l+m);
		ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

		if (j < 2*LSPAN+2) ak[j] = al[j];
		if (BLOCKSIZE > WARPSZE)	__syncthreads();
		const real sgn = 2*(j&1) - 1;	// -/+
		y0    = (it < nlat_2) ? q[im*m_inc + it] : 0.0;		// north imag (ani)
		real qer    = (it < nlat_2) ? q[(nphi-im)*m_inc + it] : 0.0;	// north real (an)
		y1    = (it < nlat_2) ? q[im*m_inc + nlat_2*2-1-it] : 0.0;	// south imag (asi)
		real qor    = (it < nlat_2) ? q[(nphi-im)*m_inc + nlat_2*2-1-it] : 0.0;	// south real (as)
		real qei = y0-qer;		qer += y0;		// ani = -qei[lane+1],   bni = qei[lane-1]
		real qoi = y1-qor;		qor += y1;		// bsi = -qoi[lane-1],   asi = qoi[lane+1];
		y0 = shfl_xor(qei, 1);	// exchange between adjacent lanes.
		y1 = shfl_xor(qoi, 1);
		reo[j] 			    = qer + qor;	// rer
		reo[BLOCKSIZE +j]   = qer - qor;	// ror
		reo[2*BLOCKSIZE +j] = sgn*(y0 - y1);	// rei
		reo[3*BLOCKSIZE +j] = sgn*(y0 + y1);	// roi

		y1 = sqrt(1.0 - cost*cost);	// sin(theta)

		y0 = mpos_scale;	// y0
		l = m - S;
		int ny = 0;
		int nsint = 0;
		do {		// sin(theta)^(m-S)		(use rescaling to avoid underflow)
			if (l&1) {
				y0 *= y1;
				ny += nsint;
				// the use of _any leads to wrong results. On KEPLER it is also slower.
				if (y0 < (accuracy+1.0/scale_factor)) {		// possible warp divergence
					ny--;
					y0 *= scale_factor;
				}
			}
			y1 *= y1;
			nsint += nsint;
			if (y1 < 1.0/scale_factor) {	// possible warp divergence
				nsint--;
				y1 *= scale_factor;
			}
		} while(l >>= 1);
		y0 *= ak[0];
		if (it < nlat_2)     y0 *= ct[it + nlat_2];		// include quadrature weights.
		y1 = ak[1]*y0*cost;


		l=m;		al+=2;
		while (l <= llim) {
			for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
				yl[k*l_inc +j]     = (ny==0) ? y0 : 0.0;
				y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
				yl[(k+1)*l_inc +j] = (ny==0) ? y1 : 0.0;
				y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
				if (ny<0) {
					if (fabs(y0) > accuracy*scale_factor + 1.0)		// possible warp divergence
					{	// rescale when value is significant
						++ny;
						y0 *= 1.0/scale_factor;
						y1 *= 1.0/scale_factor;
					}
				}
				al += 4;
			}

			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			real qlri = 0.0;	// accumulator
			// now re-assign each thread an l (transpose)
			const int ll = j / (BLOCKSIZE/LSPAN);
			const int ri = j / (BLOCKSIZE/(2*LSPAN)) % 2;	// real (0) or imag (1)
			if (ll+l <= llim) {
				for (int i=0; i<BLOCKSIZE; i+= BLOCKSIZE/(2*LSPAN)) {
				int it = j % (BLOCKSIZE/(2*LSPAN)) + i;
				qlri += reo[((ll&1)+2*ri)*BLOCKSIZE +it]   * yl[ll*l_inc +it];
				}
			}

			// reduce_add within same l must be in same warp too:
			if (BLOCKSIZE/(2*LSPAN) > WARPSZE) printf("ERROR\n");

			for (int ofs = BLOCKSIZE/(LSPAN*4); ofs > 0; ofs>>=1) {
				qlri += shfl_down(qlri, ofs, BLOCKSIZE/(LSPAN*2));
			}
			if ( ((j % (BLOCKSIZE/(2*LSPAN))) == 0) && ((l+ll)<=llim) ) {	// write result
				if (nlat_2 <= BLOCKSIZE) {		// do we need atomic add or not ?
				ql[2*(l+ll)+ri]   = qlri;
				} else {
				atomicAdd(ql+2*(l+ll)+ri, qlri);		// VERY slow atomic add on Kepler.
				}
			}
			if (j<2*LSPAN) ak[j+2] = al[j];
			if (BLOCKSIZE > WARPSZE)	__syncthreads();
			l+=LSPAN;
		}
	}
}

template<int S, int NFIELDS, typename real=double>
static void ileg_m_highllim(shtns_cfg shtns, const real* q, real *ql, const int llim, int q_dist=0, int ql_dist=0)
{
	const int lmax = shtns->lmax;
	const int mres = shtns->mres;
	const int nlat_2 = shtns->nlat_2;
	const int nphi = shtns->nphi;
	int mmax = shtns->mmax;
	real *d_alm = (sizeof(real) >= 8) ? (real*) shtns->d_alm : (real*) shtns->d_alm_f;
	real *d_ct = (sizeof(real) >= 8) ? (real*) shtns->d_ct : (real*) shtns->d_ct_f;
	hipStream_t stream = shtns->comp_stream;

	const int BLOCKSIZE = 256/NFIELDS;
	const int LSPAN_ = 8/NFIELDS;
	const int NW = 1;

	const int threadsPerBlock = BLOCKSIZE;	// can be from 32 to 1024, we should try to measure the fastest !
	const int blocksPerGrid = (nlat_2 + BLOCKSIZE*NW - 1) / (BLOCKSIZE*NW);
	if (q_dist == 0) q_dist = shtns->spat_stride;
	if (ql_dist == 0) ql_dist = shtns->nlm_stride;
	if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	for (int f=0; f<NFIELDS; f++) {
		ileg_m_highllim_kernel<BLOCKSIZE, LSPAN_, S, real><<<blocks, threads, 0, stream>>>(d_alm, d_ct, q + f*q_dist, ql + f*ql_dist, llim, nlat_2, lmax,mres, nphi, shtns->mpos_scale_analys);
	}
}


template<int S, int NFIELDS, typename real=double>
static void legendre(shtns_cfg shtns, const real *ql, real *q, const int llim, const int mmax, int spat_dist = 0)
{
	if (spat_dist == 0) spat_dist = shtns->spat_stride;
	if (mmax==0) {
		leg_m0<S,NFIELDS,real>(shtns, ql, q, llim);
	} else {
		const int limit = (sizeof(real) >= 8) ? SHT_L_RESCALE_FLY : SHT_L_RESCALE_FLY_FLOAT;
		if (llim <= limit) {
			leg_m_lowllim<S,NFIELDS,real>(shtns, ql, q, llim, mmax, spat_dist);
		} else {
			leg_m_highllim<S,NFIELDS,real>(shtns, ql, q, llim, mmax);
		}
	}
}

/// Perform SH transform on data that is already on the GPU. d_Qlm and d_Vr are pointers to GPU memory (obtained by hipMalloc() for instance)
template<int S, int NFIELDS, typename real=double>
static void ilegendre(shtns_cfg shtns, const real *q, real* ql, const int llim, int spat_dist = 0)
{
	int mmax = shtns->mmax;
	const int mres = shtns->mres;

	if (spat_dist == 0) spat_dist = shtns->spat_stride;
	hipMemsetAsync(ql, 0, sizeof(real) * NFIELDS * shtns->nlm_stride, shtns->comp_stream);		// set to zero before we start.
	if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
	if (mmax==0) {
		ileg_m0<S, NFIELDS, real>(shtns, q, ql, llim, spat_dist, shtns->nlm_stride);
	} else {
		const int limit = (sizeof(real) >= 8) ? SHT_L_RESCALE_FLY : SHT_L_RESCALE_FLY_FLOAT;
		if (llim <= limit) {
			ileg_m_lowllim<S, NFIELDS, real>(shtns, q, ql, llim, spat_dist, shtns->nlm_stride);
		} else {
			ileg_m_highllim<S, NFIELDS, real>(shtns, q, ql, llim, spat_dist, shtns->nlm_stride);
		}
	}
}
